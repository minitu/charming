#include "hip/hip_runtime.h"
#include <nvshmem.h>
#include <nvshmemx.h>

#include "scheduler.h"
#include "charming.h"
#include "chare.h"
#include "ringbuf.h"
#include "util.h"

using namespace charm;

extern __constant__ int c_my_pe;
extern __constant__ int c_n_pes;

extern __device__ spsc_ringbuf_t* mbuf;
extern __device__ size_t mbuf_size;
extern __device__ uint64_t* signal_used;
extern __device__ uint64_t* signal_addr;
extern __device__ uint64_t* signal_size;
extern __device__ uint64_t* send_addr;
extern __device__ size_t* used_indices;
extern __device__ size_t* addr_indices;

extern __device__ chare_proxy_base* chare_proxies[];

enum {
  SIGNAL_FREE = 0,
  SIGNAL_USED = 1,
  SIGNAL_CLUP = 2
};

__device__ envelope* charm::create_envelope(msgtype type, size_t msg_size) {
  // Reserve space for this message in message buffer
  ringbuf_off_t mret = spsc_ringbuf_acquire(mbuf, msg_size);
  assert(mret != -1 && mret < mbuf_size);

  // Create envelope
  return new (mbuf->addr(mret)) envelope(type, msg_size, c_my_pe);
}

__device__ void charm::send_msg(envelope* env, size_t msg_size, int dst_pe) {
  // Message is ready to be sent
  spsc_ringbuf_produce(mbuf);

  // Obtain a message index for the target PE and set the corresponding used signal
  size_t offset = MSG_IN_FLIGHT_MAX * dst_pe;
  uint64_t* my_signal_used = signal_used + offset;
  size_t msg_idx = nvshmem_uint64_wait_until_any(my_signal_used, MSG_IN_FLIGHT_MAX,
      nullptr, NVSHMEM_CMP_EQ, SIGNAL_FREE);
#ifdef DEBUG
  assert(msg_idx != SIZE_MAX);
  printf("PE %d sending message request to PE %d, local message index %llu (global %llu), addr %p, size %llu\n",
      c_my_pe, dst_pe, msg_idx, offset + msg_idx, env, msg_size);
#endif
  nvshmemx_signal_op(my_signal_used + msg_idx, SIGNAL_USED, NVSHMEM_SIGNAL_SET, c_my_pe);

  // Store source buffer address for later cleanup
  send_addr[offset + msg_idx] = (uint64_t)env;

  // Send address of source buffer and message size
  // TODO: Send as one 128-bit buffer?
  offset = MSG_IN_FLIGHT_MAX * c_my_pe;
  uint64_t* my_signal_addr = signal_addr + offset;
  uint64_t* my_signal_size = signal_size + offset;
  nvshmemx_signal_op(my_signal_addr + msg_idx, (uint64_t)env, NVSHMEM_SIGNAL_SET, dst_pe);
  nvshmemx_signal_op(my_signal_size + msg_idx, (uint64_t)msg_size, NVSHMEM_SIGNAL_SET, dst_pe);
}

__device__ void charm::send_dummy_msg(int dst_pe) {
  size_t msg_size = envelope::alloc_size(0);
  envelope* env = create_envelope(msgtype::dummy, msg_size);

  send_msg(env, msg_size, dst_pe);
}

__device__ void charm::send_reg_msg(int chare_id, int chare_idx, int ep_id,
                                    void* buf, size_t payload_size, int dst_pe) {
  size_t msg_size = envelope::alloc_size(sizeof(regular_msg) + payload_size);
  envelope* env = create_envelope(msgtype::regular, msg_size);

  regular_msg* msg = new ((char*)env + sizeof(envelope)) regular_msg(chare_id, chare_idx, ep_id);

  // Fill in payload (from regular GPU memory to NVSHMEM symmetric memory)
  if (payload_size > 0) {
    memcpy((char*)msg + sizeof(regular_msg), buf, payload_size);
  }

  send_msg(env, msg_size, dst_pe);
}

__device__ void charm::send_begin_term_msg(int dst_pe) {
  size_t msg_size = envelope::alloc_size(0);
  envelope* env = create_envelope(msgtype::begin_terminate, msg_size);

  send_msg(env, msg_size, dst_pe);
}

__device__ void charm::send_do_term_msg(int dst_pe) {
  size_t msg_size = envelope::alloc_size(0);
  envelope* env = create_envelope(msgtype::do_terminate, msg_size);

  send_msg(env, msg_size, dst_pe);
}

__device__ __forceinline__ ssize_t next_msg(void* addr, bool& begin_term_flag,
                                            bool& do_term_flag) {
  static int dummy_cnt = 0;
  static clock_value_t start;
  static clock_value_t end;
  envelope* env = (envelope*)addr;
#ifdef DEBUG
  printf("PE %d received msg type %d size %llu from PE %d\n",
         nvshmem_my_pe(), env->type, env->size, env->src_pe);
#endif

  if (env->type == msgtype::dummy) {
    // Dummy message
    if (dummy_cnt == 0) {
      start = clock64();
    } else if (dummy_cnt == DUMMY_ITERS-1) {
      end = clock64();
      printf("Receive avg clocks: %lld\n", (end - start) / DUMMY_ITERS);
    }
    dummy_cnt++;
  } else if (env->type == msgtype::create) {
    // Creation message
    create_msg* msg = (create_msg*)((char*)env + sizeof(envelope));
#ifdef DEBUG
    printf("PE %d creation msg chare ID %d, n_local %d, n_total %d, start idx %d, end idx %d\n",
           nvshmem_my_pe(), msg->chare_id, msg->n_local, msg->n_total, msg->start_idx, msg->end_idx);
#endif
    chare_proxy_base*& chare_proxy = chare_proxies[msg->chare_id];
    chare_proxy->alloc(msg->n_local, msg->n_total, msg->start_idx, msg->end_idx);
    char* tmp = (char*)msg + sizeof(create_msg);
    chare_proxy->store_loc_map(tmp);
    tmp += sizeof(int) * msg->n_total;
    for (int i = 0; i < msg->n_local; i++) {
      chare_proxy->unpack(tmp, i);
    }
  } else if (env->type == msgtype::regular) {
    // Regular message
    regular_msg* msg = (regular_msg*)((char*)env + sizeof(envelope));
#ifdef DEBUG
    printf("PE %d regular msg chare ID %d chare idx %d EP ID %d\n", nvshmem_my_pe(), msg->chare_id, msg->chare_idx, msg->ep_id);
#endif
    chare_proxy_base*& chare_proxy = chare_proxies[msg->chare_id];
    void* payload = (char*)msg + sizeof(regular_msg);
    // TODO: Copy payload?
    chare_proxy->call(msg->chare_idx, msg->ep_id, payload);
  } else if (env->type == msgtype::begin_terminate) {
    // Should only be received by PE 0
    assert(my_pe() == 0);
    // Begin termination message
#ifdef DEBUG
    printf("PE %d begin terminate msg\n", nvshmem_my_pe());
#endif
    if (!begin_term_flag) {
      for (int i = 0; i < n_pes(); i++) {
        send_do_term_msg(i);
      }
      begin_term_flag = true;
    }
  } else if (env->type == msgtype::do_terminate) {
    // Do termination message
#ifdef DEBUG
    printf("PE %d do terminate msg\n", nvshmem_my_pe());
#endif
    do_term_flag = true;
  }

  return env->size;
}

__device__ __forceinline__ void recv_msg(bool& begin_term_flag, bool &do_term_flag) {
  // Check if there are any message requests
  size_t count = nvshmem_uint64_test_some(signal_addr, MSG_IN_FLIGHT_MAX * c_n_pes,
      addr_indices, nullptr, NVSHMEM_CMP_GT, 0);
  if (count > 0) {
    for (size_t i = 0; i < count; i++) {
      // Obtain information about this message request
      size_t msg_idx = addr_indices[i];
      uint64_t src_addr = nvshmem_signal_fetch(signal_addr + msg_idx);
      uint64_t src_size = nvshmem_signal_fetch(signal_size + msg_idx);
      int src_pe = msg_idx / MSG_IN_FLIGHT_MAX;
      msg_idx -= MSG_IN_FLIGHT_MAX * src_pe;
#ifdef DEBUG
      printf("PE %d received message request from PE %d, local message index %llu "
          "(global %llu), addr %p, size %llu\n", c_my_pe, src_pe, msg_idx,
          MSG_IN_FLIGHT_MAX * src_pe + msg_idx, (void*)src_addr, src_size);
#endif

      // Reserve space for incoming message
      ringbuf_off_t mret = spsc_ringbuf_acquire(mbuf, src_size);
      assert(mret != -1 && mret < mbuf_size);

      // Perform a get operation to fetch the message
      // TODO: Make asynchronous
      nvshmem_char_get((char*)mbuf->addr(mret), (char*)src_addr, src_size, src_pe);

      // Process message
      next_msg(mbuf->addr(mret), begin_term_flag, do_term_flag);

      // Clear message request
      nvshmemx_signal_op(signal_addr + MSG_IN_FLIGHT_MAX * src_pe + msg_idx,
          SIGNAL_FREE, NVSHMEM_SIGNAL_SET, c_my_pe);

      // Notify sender that message is ready for cleanup
      nvshmemx_signal_op(signal_used + MSG_IN_FLIGHT_MAX * c_my_pe + msg_idx,
          SIGNAL_CLUP, NVSHMEM_SIGNAL_SET, src_pe);
    }

    // Reset indices array for next use
    memset(addr_indices, 0, MSG_IN_FLIGHT_MAX * c_n_pes * sizeof(size_t));
  }

  // Clean up completed messages
  count = nvshmem_uint64_test_some(signal_used, MSG_IN_FLIGHT_MAX * c_n_pes,
      used_indices, nullptr, NVSHMEM_CMP_EQ, SIGNAL_CLUP);
  if (count > 0) {
    for (size_t i = 0; i < count; i++) {
      size_t msg_idx = used_indices[i];
#ifdef DEBUG
      printf("PE %d cleaning up global message index %llu\n", c_my_pe, msg_idx);
#endif
      uint64_t src_addr = send_addr[msg_idx];
      // TODO: Free message
      // Need mapping from message index to address
      /*
      size_t len, off;
      len = spsc_ringbuf_consume(mbuf, &off);
      spsc_ringbuf_release(mbuf, len);
      */

      // Reset signal to SIGNAL_FREE
      nvshmemx_signal_op(signal_used + msg_idx, SIGNAL_FREE,
          NVSHMEM_SIGNAL_SET, c_my_pe);
    }

    // Reset indices array for next use
    memset(used_indices, 0, MSG_IN_FLIGHT_MAX * c_n_pes * sizeof(size_t));
  }
}

__global__ void charm::scheduler(int argc, char** argv, size_t* argvs) {
  if (!blockIdx.x && !threadIdx.x) {
    bool begin_term_flag = false;
    bool do_term_flag = false;

    // Register user chares and entry methods on all PEs
    chare_proxy_cnt = 0;
    register_chares();

    // Initialize message queue
    spsc_ringbuf_init(mbuf, mbuf_size);

    nvshmem_barrier_all();

    if (c_my_pe == 0) {
      // Execute user's main function
      main(argc, argv, argvs);
    }

    nvshmem_barrier_all(); // FIXME: No need?

    // Receive messages and terminate
    do {
      recv_msg(begin_term_flag, do_term_flag);
    } while (!do_term_flag);

#if DEBUG
    printf("PE %d terminating...\n", c_my_pe);
#endif
  }
}
