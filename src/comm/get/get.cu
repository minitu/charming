#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#include "charming.h"
#include "common.h"
#include "kernel.h"
#include "message.h"
#include "comm.h"
#include "scheduler.h"
#include "ringbuf.h"
#include "composite.h"
#include "heap.h"
#include "util.h"

// Use custom thread block extensions to NVSHMEM
#define NVSHMEM_BLOCK_EXT

// Use thread block extension of NVSHMEM for communication
// Turned off for now as it shows identical performance
//#define NVSHMEM_BLOCK_COMM

#define MBUF_PE_SIZE 8388608 // 8MB per PE
#define MBUF_CE_SIZE 134217728 // 128MB per CE

/*
// Maximum number of messages that are allowed to be in flight per pair of PEs
#define REMOTE_MSG_COUNT_MAX 4
// Maximum number of messages that can be stored in the local message queue
#define LOCAL_MSG_COUNT_MAX 128
*/
#define LOCAL_MSG_MAX 4
#define REMOTE_MSG_MAX 4

using namespace charm;

typedef unsigned long long int atomic64_t;

extern hipStream_t stream;

// Managed memory (actual data may reside elsewhere)
__managed__ void* nvshmem_buf; // NVSHMEM

__managed__ ringbuf_t* mbuf; // Managed

__managed__ volatile int* send_status_local; // Global
__managed__ uint64_t* send_status_remote; // NVSHMEM
__managed__ size_t* send_status_remote_idx; // Global

__managed__ uint64_t* send_comp_local; // Global
__managed__ uint64_t* send_comp_remote; // Global

__managed__ volatile atomic64_t* recv_comp_local; // Global
__managed__ uint64_t* recv_comp_remote; // NVSHMEM
__managed__ size_t* recv_comp_remote_idx; // Global

__managed__ composite_t* heap_buf; // Global

// GPU shared memory
extern __shared__ uint64_t s_mem[];

enum {
  SIGNAL_FREE = 0,
  SIGNAL_USED = 1,
  SIGNAL_CLUP = 2
};

void charm::comm_init_host(int n_sms, int n_pes, int n_ces, int n_clusters_dev,
    int n_pes_cluster, int n_ces_cluster) {
  // Allocate NVSHMEM message buffer
  size_t mbuf_cluster_size = MBUF_PE_SIZE * n_pes_cluster + MBUF_CE_SIZE * n_ces_cluster;
  nvshmem_buf = nvshmem_malloc(mbuf_cluster_size * n_clusters_dev);
  assert(nvshmem_buf);
  hipMallocManaged(&mbuf, sizeof(ringbuf_t) * n_sms);
  assert(mbuf);
  ringbuf_t* cur_mbuf = mbuf;
  size_t start_offset = 0;
  for (int i = 0; i < n_sms; i++) {
    int cluster_size = n_pes_cluster + n_ces_cluster;
    int rank_in_cluster = i % cluster_size;
    bool is_pe = rank_in_cluster < n_pes_cluster;
    size_t mbuf_size = is_pe ? MBUF_PE_SIZE : MBUF_CE_SIZE;
    cur_mbuf->init(nvshmem_buf, start_offset, mbuf_size);
    start_offset += mbuf_size;
    cur_mbuf++;
  }

  // Allocate data structures
  int n_ces_dev = n_ces_cluster * n_clusters_dev;
  size_t local_count = LOCAL_MSG_MAX * n_sms * n_sms;
  size_t remote_count = REMOTE_MSG_MAX * n_ces * n_ces_dev;
  size_t status_local_size = sizeof(int) * local_count;
  size_t status_remote_size = sizeof(uint64_t) * remote_count;
  size_t idx_size = sizeof(size_t) * remote_count;
  size_t comp_local_size = sizeof(atomic64_t) * local_count;
  size_t comp_remote_size = sizeof(atomic64_t) * remote_count;
  size_t heap_size = sizeof(composite_t) * local_count * 2;
  assert(sizeof(atomic64_t) == sizeof(uint64_t));
  hipMalloc(&send_status_local, status_local_size);
  send_status_remote = (uint64_t*)nvshmem_malloc(status_remote_size);
  hipMalloc(&send_status_remote_idx, idx_size);
  hipMalloc(&send_comp_local, comp_local_size);
  hipMalloc(&send_comp_remote, comp_remote_size);
  hipMalloc(&recv_comp_local, comp_local_size);
  recv_comp_remote = (uint64_t*)nvshmem_malloc(comp_remote_size);
  hipMalloc(&recv_comp_remote_idx, idx_size);
  hipMalloc(&heap_buf, heap_size);
  assert(send_status_local && send_status_remote && send_status_remote_idx
      && send_comp_local && send_comp_remote && recv_comp_local
      && recv_comp_remote && recv_comp_remote_idx && heap_buf);

  // Clear data structures
  hipMemsetAsync((void*)send_status_local, 0, status_local_size, stream);
  hipMemsetAsync((void*)send_status_remote, 0, status_remote_size, stream);
  hipMemsetAsync((void*)send_status_remote_idx, 0, idx_size, stream);
  hipMemsetAsync((void*)send_comp_local, 0, comp_local_size, stream);
  hipMemsetAsync((void*)send_comp_remote, 0, comp_remote_size, stream);
  hipMemsetAsync((void*)recv_comp_local, 0, comp_local_size, stream);
  hipMemsetAsync((void*)recv_comp_remote, 0, comp_remote_size, stream);
  hipMemsetAsync((void*)recv_comp_remote_idx, 0, idx_size, stream);
  hipMemsetAsync((void*)heap_buf, 0, heap_size, stream);
  hipStreamSynchronize(stream);
  cuda_check_error();
}

void charm::comm_fini_host() {
  // Free NVSHMEM message buffer
  nvshmem_free(nvshmem_buf);

  // Free data structures
  hipFree((void*)mbuf);
  hipFree((void*)send_status_local);
  nvshmem_free(send_status_remote);
  hipFree((void*)send_status_remote_idx);
  hipFree((void*)send_comp_local);
  hipFree((void*)send_comp_remote);
  hipFree((void*)recv_comp_local);
  nvshmem_free(recv_comp_remote);
  hipFree((void*)recv_comp_remote_idx);
  hipFree((void*)heap_buf);
}

// Single-threaded
__device__ void charm::comm::init() {
  // Initialize min-heap
  int comp_count = LOCAL_MSG_MAX * c_n_sms * 2;
  composite_t* my_heap_buf = heap_buf + comp_count * blockIdx.x;
  addr_heap.init(my_heap_buf, comp_count);

  sent_term_flag = false;
  begin_term_flag = false;
  do_term_flag = false;

  if (!s_mem[s_idx::is_pe]) {
    // Store local ranks and count of child PEs for this CE
    child_count = 0;
    child_local_ranks = new int[c_n_pes_cluster];
    assert(child_local_ranks);

    int my_cluster = blockIdx.x / c_cluster_size;
    int my_rank_in_cluster = blockIdx.x % c_cluster_size;
    int start_local_rank = my_cluster * c_cluster_size;
    for (int i = 0; i < c_n_pes_cluster; i++) {
      int ce_rank_in_cluster = i % c_n_ces_cluster + c_n_pes_cluster;
      if (ce_rank_in_cluster == my_rank_in_cluster) {
        child_local_ranks[child_count++] = start_local_rank + i;
      }
    }
  }
}

__device__ __forceinline__ int find_signal_single(volatile int* status,
    int count, int old_val, int new_val, bool loop) {
  int idx = INT_MAX;

  // Look for desired signal
  do {
    for (int i = 0; i < count; i++) {
      if (status[i] == old_val) {
        idx = i;
      }
      __threadfence();

      if (idx != INT_MAX) break;
    }
  } while (loop && idx == INT_MAX);

  // Update signal if necessary
  if (idx != INT_MAX && old_val != new_val) {
    int ret = atomicCAS((int*)&status[idx], old_val, new_val);
    assert(ret == old_val);
  }

  return idx;
}

__device__ __forceinline__ int find_signal_block(volatile int* status,
    int count, int old_val, int new_val, bool loop) {
  __shared__ volatile int idx;
  if (threadIdx.x == 0) idx = INT_MAX;
  __syncthreads();

  // Look for desired signal
  do {
    for (int i = threadIdx.x; i < count; i += blockDim.x) {
      if (status[i] == old_val) {
        atomicMin_block((int*)&idx, i);
      }
      __threadfence();
    }
  } while (loop && idx == INT_MAX);
  __syncthreads();

  // Update signal if necessary
  if (idx != INT_MAX && old_val != new_val && threadIdx.x == 0) {
    int ret = atomicCAS((int*)&status[idx], old_val, new_val);
    assert(ret == old_val);
  }
  __syncthreads();

  return idx;
}

__device__ __forceinline__ atomic64_t find_msg_single(volatile atomic64_t* comps, int& idx) {
  idx = INT_MAX;
  atomic64_t comp = 0;

  // Look for a valid message (traverse once)
  for (int i = 0; i < LOCAL_MSG_MAX * c_n_sms; i++) {
    comp = comps[i];
    __threadfence();

    // If a message is found, reset message address to zero
    if (comp) {
      idx = i;
      atomic64_t ret = atomicCAS((atomic64_t*)&comps[idx], comp, 0);
      assert(ret == comp);
      break;
    }
  }

  return comp;
}

__device__ __forceinline__ atomic64_t find_msg_block(volatile atomic64_t* comps, int& ret_idx) {
  __shared__ volatile int idx;
  __shared__ atomic64_t comp;
  if (threadIdx.x == 0) {
    idx = INT_MAX;
    comp = 0;
  }
  __syncthreads();

  // Look for a valid message (traverse once)
  for (int i = threadIdx.x; i < LOCAL_MSG_MAX * c_n_sms; i += blockDim.x) {
    if (comps[i] != 0) {
      atomicMin_block((int*)&idx, i);
    }
    __threadfence();
  }
  __syncthreads();
  ret_idx = idx;

  // If a message is found
  if (idx != INT_MAX && threadIdx.x == 0) {
    comp = (atomic64_t)comps[idx];
    __threadfence();

    // Reset message address to zero
    atomic64_t ret = atomicCAS((atomic64_t*)&comps[idx], comp, 0);
    assert(ret == comp);
  }
  __syncthreads();

  return comp;
}

__device__ void charm::comm::process_local() {
  int dst_local_rank = blockIdx.x;
  bool is_pe = (s_mem[s_idx::is_pe] == 1);
#ifdef DEBUG
  int dst_elem = is_pe ? s_mem[s_idx::my_pe] : s_mem[s_idx::my_ce];
#endif

  // Look for valid message addresses
  volatile atomic64_t* recv_comp = recv_comp_local + LOCAL_MSG_MAX * c_n_sms * dst_local_rank;
  int msg_idx;
  composite_t comp((uint64_t)find_msg_block(recv_comp, msg_idx));

  if (comp.data) {
    int src_local_rank = msg_idx / LOCAL_MSG_MAX;
    int clup_idx = msg_idx % LOCAL_MSG_MAX;
    ringbuf_t* dst_mbuf = mbuf + dst_local_rank;
    envelope* env = (envelope*)dst_mbuf->addr(comp.offset());
    if (threadIdx.x == 0) {
      PDEBUG("%s %d receiving local message (env %p, msgtype %d, size %llu) "
          "from local rank %d at index %d\n",
          is_pe ? "PE" : "CE", dst_elem, env, env->type, env->size,
          src_local_rank, clup_idx);
    }
    __syncthreads();

    // Process message in parallel
    msgtype type = is_pe ?
      process_msg_pe(env, comp.offset(), begin_term_flag, do_term_flag)
      : process_msg_ce(env, comp.offset(), sent_term_flag, begin_term_flag,
          do_term_flag);

    // Signal sender for cleanup
    if (threadIdx.x == 0 && type != msgtype::user) {
      PDEBUG("%s %d process_local signal cleanup (env %p, msgtype %d, size %llu) "
          "to local rank %d at index %d\n",
          is_pe ? "PE" : "CE", dst_elem, env, env->type, env->size,
          src_local_rank, clup_idx);

      volatile int* src_send_status = send_status_local
        + LOCAL_MSG_MAX * c_n_sms * src_local_rank + LOCAL_MSG_MAX * dst_local_rank;
#ifndef NO_CLEANUP
      int signal = SIGNAL_CLUP;
#else
      int signal = SIGNAL_FREE;
#endif
      int ret = atomicCAS((int*)&src_send_status[clup_idx], SIGNAL_USED, signal);
      assert(ret == SIGNAL_USED);
    }
    __syncthreads();
  }
}

__device__ void charm::comm::process_remote() {
  int dst_local_rank = blockIdx.x;
  int dst_ce = s_mem[s_idx::my_ce];
  int dst_ce_dev = get_ce_in_dev(dst_ce);
  int dst_dev = get_dev_from_ce(dst_ce);

  // Check if there are any incoming messages
  uint64_t* recv_comp = recv_comp_remote + (REMOTE_MSG_MAX * c_n_ces) * dst_ce_dev;
  size_t* recv_comp_idx = recv_comp_remote_idx + (REMOTE_MSG_MAX * c_n_ces) * dst_ce_dev;
  size_t count = 0;
#ifdef NVSHMEM_BLOCK_EXT
  count = nvshmem_uint64_test_some_block(recv_comp, REMOTE_MSG_MAX * c_n_ces,
      recv_comp_idx, NVSHMEM_CMP_GT, 0);
#else
  if (threadIdx.x == 0) {
    count = nvshmem_uint64_test_some(recv_comp, REMOTE_MSG_MAX * c_n_ces,
        recv_comp_idx, nullptr, NVSHMEM_CMP_GT, 0);
    s_mem[s_idx::size] = (uint64_t)count;
  }
  __syncthreads();
  count = (size_t)s_mem[s_idx::size];
#endif

  if (count > 0) {
    ringbuf_t* dst_mbuf = mbuf + dst_local_rank;;
    size_t found_idx;
    uint64_t data;
    size_t src_offset;
    size_t msg_size;
    int src_ce;
    int src_ce_dev;
    int src_dev;
    size_t msg_idx;
    size_t dst_offset;
    void* dst_addr = nullptr;
    void* src_addr = nullptr;

    for (size_t i = 0; i < count; i++) {
      if (threadIdx.x == 0) {
        // Obtain information about this message
        found_idx = recv_comp_idx[i];
        data = nvshmem_signal_fetch(recv_comp + found_idx);
        composite_t src_composite(data);
        src_offset = src_composite.offset();
        msg_size = src_composite.size();
        src_ce = found_idx / REMOTE_MSG_MAX;
        msg_idx = found_idx % REMOTE_MSG_MAX;

        // Reserve space for incoming message
        bool success = dst_mbuf->acquire(msg_size, dst_offset);
        if (!success) {
          PERROR("CE %d: Not enough space in message buffer\n", dst_ce);
          assert(false);
        }
        PDEBUG("CE %d acquired space for incoming remote message: offset %llu, size %llu\n",
            dst_ce, dst_offset, msg_size);

        // Perform a get operation to fetch the message
        // TODO: Make asynchronous
        dst_addr = dst_mbuf->addr(dst_offset);
        src_addr = dst_mbuf->addr(src_offset);
        src_ce_dev = get_ce_in_dev(src_ce);
        src_dev = get_dev_from_ce(src_ce);
        s_mem[s_idx::dst] = (uint64_t)dst_addr;
        s_mem[s_idx::offset] = (uint64_t)dst_offset;
#ifdef NVSHMEM_BLOCK_COMM
        s_mem[s_idx::src] = (uint64_t)src_addr;
        s_mem[s_idx::size] = (uint64_t)msg_size;
        s_mem[s_idx::dev] = (uint64_t)src_dev;
#else
        nvshmem_char_get((char*)dst_addr, (char*)src_addr, msg_size, src_dev);
#endif
        PDEBUG("CE %d remote get: src offset %llu, dst offset %llu, "
            "size %llu, src CE %d, idx %llu\n", dst_ce, src_offset, dst_offset,
            msg_size, src_ce, msg_idx);
      }
      __syncthreads();
      dst_addr = (void*)s_mem[s_idx::dst];
      dst_offset = (size_t)s_mem[s_idx::offset];
#ifdef NVSHMEM_BLOCK_COMM
      src_addr = (void*)s_mem[s_idx::src];
      msg_size = (size_t)s_mem[s_idx::size];
      src_dev = (int)s_mem[s_idx::dev];
      nvshmemx_char_get_block((char*)dst_addr, (char*)src_addr, msg_size, src_dev);
#endif

      // Process message in parallel
      msgtype type = process_msg_ce(dst_addr, dst_offset, sent_term_flag,
          begin_term_flag, do_term_flag);

      if (threadIdx.x == 0) {
        // Clear message request
        // FIXME: Need fence after?
        nvshmemx_signal_op(recv_comp + found_idx, SIGNAL_FREE, NVSHMEM_SIGNAL_SET,
            dst_dev);

        uint64_t* src_send_status = send_status_remote
          + (REMOTE_MSG_MAX * c_n_ces) * src_ce_dev
          + REMOTE_MSG_MAX * dst_ce;
        int signal = SIGNAL_FREE;
#ifndef NO_CLEANUP
        // Store composite to be cleared from memory
        composite_t dst_composite(dst_offset, msg_size);
        // Forwarded message should not be freed here
        // It will be freed as a local message once it arrives on the destination PE
        if (type != msgtype::forward) {
          addr_heap.push(dst_composite);
        }
        signal = (type == msgtype::user) ? SIGNAL_FREE : SIGNAL_CLUP;
        PDEBUG("CE %d process_remote signal cleanup & push to heap: signal %d, "
            "offset %llu, size %llu, src CE %d, idx %llu\n", dst_ce, signal,
            dst_composite.offset(), dst_composite.size(), src_ce, msg_idx);
#endif
        // Notify sender that message has been delivered
        nvshmemx_signal_op(src_send_status + msg_idx, signal, NVSHMEM_SIGNAL_SET,
            src_dev);
      }
      __syncthreads();
    }

    // Reset indices array for next use
    memset_kernel(recv_comp_idx, 0, REMOTE_MSG_MAX * c_n_ces * sizeof(size_t));
  }
}

__device__ void charm::comm::cleanup_local() {
  int local_rank = blockIdx.x;
  volatile int* send_status = send_status_local + LOCAL_MSG_MAX * c_n_sms * local_rank;
  int clup_idx = find_signal_block(send_status, LOCAL_MSG_MAX * c_n_sms,
      SIGNAL_CLUP, SIGNAL_FREE, false);

  // If a message needs to be cleaned up, add composite to min-heap
  if (clup_idx != INT_MAX && threadIdx.x == 0) {
    uint64_t* send_comp = send_comp_local + LOCAL_MSG_MAX * c_n_sms * local_rank;
    composite_t comp(send_comp[clup_idx]);
    addr_heap.push(comp);
    PDEBUG("%s %d cleanup_local push to heap: "
        "offset %llu, size %llu, dst local rank %d, msg idx %d\n",
        s_mem[s_idx::is_pe] ? "PE" : "CE",
        s_mem[s_idx::is_pe] ? (int)s_mem[s_idx::my_pe] : (int)s_mem[s_idx::my_ce],
        comp.offset(), comp.size(), clup_idx / LOCAL_MSG_MAX,
        clup_idx % LOCAL_MSG_MAX);
  }
  __syncthreads();
}

__device__ void charm::comm::cleanup_remote() {
  int my_ce_dev = get_ce_in_dev(s_mem[s_idx::my_ce]);
  int my_dev = get_dev_from_ce(s_mem[s_idx::my_ce]);

  // Check for messages that have been delivered to the destination PE
  uint64_t* send_status = send_status_remote + (REMOTE_MSG_MAX * c_n_ces) * my_ce_dev;
  uint64_t* send_status_idx = send_status_remote_idx + (REMOTE_MSG_MAX * c_n_ces) * my_ce_dev;
  size_t count = 0;
#ifdef NVSHMEM_BLOCK_EXT
  count = nvshmem_uint64_test_some_block(send_status, REMOTE_MSG_MAX * c_n_ces,
      send_status_idx, NVSHMEM_CMP_EQ, SIGNAL_CLUP);
#else
  if (threadIdx.x == 0) {
    count = nvshmem_uint64_test_some(send_status, REMOTE_MSG_MAX * c_n_ces,
        send_status_idx, nullptr, NVSHMEM_CMP_EQ, SIGNAL_CLUP);
    s_mem[s_idx::size] = (uint64_t)count;
  }
  __syncthreads();
  count = (size_t)s_mem[s_idx::size];
#endif

  // Push composites to min-heap for cleanup
  if (count > 0) {
    if (threadIdx.x == 0) {
      uint64_t* send_comp = send_comp_remote + (REMOTE_MSG_MAX * c_n_ces) * my_ce_dev;

      for (size_t i = 0; i < count; i++) {
        size_t found_idx = send_status_idx[i];
        composite_t src_composite(send_comp[found_idx]);
        addr_heap.push(src_composite);
        PDEBUG("CE %d cleanup_remote push to heap: offset %llu, size %llu, "
            "dst CE %llu, idx %llu, found_idx %llu\n", (int)s_mem[s_idx::my_ce],
            src_composite.offset(), src_composite.size(),
            found_idx / REMOTE_MSG_MAX, found_idx % REMOTE_MSG_MAX, found_idx);

        // Reset signal to SIGNAL_FREE
        nvshmemx_signal_op(send_status + found_idx, SIGNAL_FREE,
            NVSHMEM_SIGNAL_SET, my_dev);
      }
    }
    __syncthreads();

    // Reset indices array for next use
    memset_kernel(send_status_idx, 0, REMOTE_MSG_MAX * c_n_ces * sizeof(size_t));
  }
}

__device__ void charm::comm::cleanup_heap() {
  // Check min-heap and free messages
  if (threadIdx.x == 0) {
    int local_rank = blockIdx.x;
    composite_t top;
    size_t clup_offset;
    size_t clup_size;
    ringbuf_t* my_mbuf = mbuf + local_rank;
    while (true) {
      top = addr_heap.top();
      if (top.data == UINT64_MAX) break;

      clup_offset = top.offset();
      clup_size = top.size();
      if ((clup_offset == my_mbuf->start_offset + my_mbuf->read) && clup_size > 0) {
        bool success = my_mbuf->release(clup_size);
        if (!success) {
          PERROR("%s %d failed to release message: offset %llu, size %llu\n",
              s_mem[s_idx::is_pe] ? "PE" : "CE",
              s_mem[s_idx::is_pe] ? (int)s_mem[s_idx::my_pe] : (int)s_mem[s_idx::my_ce],
              clup_offset, clup_size);
          my_mbuf->print();
          assert(false);
        }
        addr_heap.pop();
        PDEBUG("%s %d releasing message: offset %llu, size %llu\n",
            s_mem[s_idx::is_pe] ? "PE" : "CE",
            s_mem[s_idx::is_pe] ? (int)s_mem[s_idx::my_pe] : (int)s_mem[s_idx::my_ce],
            clup_offset, clup_size);
      } else break;
    }
  }
  __syncthreads();
}

/*
__device__ void charm::message::alloc(int idx, int ep, size_t size) {
  size_t msg_size = envelope::alloc_size(sizeof(regular_msg) + size);
  env = create_envelope(msgtype::user, msg_size, &offset);
}

__device__ void charm::message::free() {
  // TODO
}
*/

// Single-threaded
__device__ envelope* charm::create_envelope(msgtype type, size_t payload_size,
    size_t& offset) {
  size_t msg_size = envelope::alloc_size(type, payload_size);

  // Reserve space for this message in message buffer
  bool is_pe = (s_mem[s_idx::is_pe] == 1);
  int my_elem = is_pe ? s_mem[s_idx::my_pe] : s_mem[s_idx::my_ce];
  ringbuf_t* src_mbuf = mbuf + blockIdx.x;
  bool success = src_mbuf->acquire(msg_size, offset);
  if (!success) {
    PERROR("%s %d: Not enough space in message buffer\n",
        is_pe ? "PE" : "CE", my_elem);
    src_mbuf->print();
    assert(false);
  }
  PDEBUG("%s %d acquired message: offset %llu, size %llu\n",
      is_pe ? "PE" : "CE", my_elem, offset, msg_size);

  // Create envelope
  return new (src_mbuf->addr(offset)) envelope(type, msg_size);
}

__device__ void charm::send_local_msg(envelope* env, size_t offset, int dst_local_rank) {
  int src_local_rank = blockIdx.x;
  volatile int* send_status = send_status_local
    + LOCAL_MSG_MAX * c_n_sms * src_local_rank + LOCAL_MSG_MAX * dst_local_rank;

  // Find and reserve free message index
  int free_idx = find_signal_block(send_status, LOCAL_MSG_MAX, SIGNAL_FREE,
      SIGNAL_USED, true);

  if (threadIdx.x == 0) {
    PDEBUG("%s %d sending local message: dst local rank %d, "
        "index %d, env %p, msgtype %d, size %llu\n", s_mem[s_idx::is_pe] ? "PE" : "CE",
        s_mem[s_idx::is_pe] ? (int)s_mem[s_idx::my_pe] : (int)s_mem[s_idx::my_ce],
        dst_local_rank, free_idx, env, env->type, env->size);

    // Atomically store composite in receiver
    volatile atomic64_t* recv_comp = recv_comp_local
      + LOCAL_MSG_MAX * c_n_sms * dst_local_rank + LOCAL_MSG_MAX * src_local_rank;
    composite_t comp(offset, env->size);
    atomic64_t ret = atomicCAS((atomic64_t*)&recv_comp[free_idx], 0,
        (atomic64_t)comp.data);
    assert(ret == 0);

#ifndef NO_CLEANUP
    // Store composite for later cleanup
    uint64_t* send_comp = send_comp_local
      + LOCAL_MSG_MAX * c_n_sms * src_local_rank + LOCAL_MSG_MAX * dst_local_rank;
    send_comp[free_idx] = comp.data;
#endif
  }
  __syncthreads();
}

__device__ void charm::send_remote_msg(envelope* env, size_t offset, int dst_ce) {
  if (threadIdx.x == 0) {
    int src_ce = s_mem[s_idx::my_ce];
    int src_ce_dev = get_ce_in_dev(src_ce);
    int dst_ce_dev = get_ce_in_dev(dst_ce);
    int dst_dev = get_dev_from_ce(dst_ce);

    // Obtain a message index for the target CE and set signal to used
    uint64_t* send_status = send_status_remote + (REMOTE_MSG_MAX * c_n_ces) * src_ce_dev
      + REMOTE_MSG_MAX * dst_ce;
    size_t msg_idx = nvshmem_uint64_wait_until_any(send_status, REMOTE_MSG_MAX,
        nullptr, NVSHMEM_CMP_EQ, SIGNAL_FREE);
    nvshmemx_signal_op(send_status + msg_idx, SIGNAL_USED, NVSHMEM_SIGNAL_SET, c_my_dev);

    // Send composite
    uint64_t* recv_comp = recv_comp_remote + (REMOTE_MSG_MAX * c_n_ces) * dst_ce_dev
      + REMOTE_MSG_MAX * src_ce;
    composite_t src_composite(offset, env->size);
    nvshmemx_signal_op(recv_comp + msg_idx, src_composite.data, NVSHMEM_SIGNAL_SET,
        dst_dev);
    PDEBUG("CE %d sending remote message: offset %llu, size %llu, dst CE %d, idx %llu\n",
        src_ce, offset, env->size, dst_ce, msg_idx);

#ifndef NO_CLEANUP
    // Store source composite for later cleanup
    uint64_t* send_comp = send_comp_remote + (REMOTE_MSG_MAX * c_n_ces) * src_ce_dev
      + REMOTE_MSG_MAX * dst_ce;
    send_comp[msg_idx] = src_composite.data;
#endif
  }
  __syncthreads();
}

__device__ void charm::send_reg_msg(int chare_id, int chare_idx, int ep_id,
    void* buf, size_t payload_size, int dst_pe) {
  if (threadIdx.x == 0) {
    int src_dev = get_dev_from_pe(s_mem[s_idx::my_pe]);
    int dst_dev = get_dev_from_pe(dst_pe);
    envelope* env = nullptr;
    if (src_dev == dst_dev) {
      // Need to send to another PE on same device
      env = create_envelope(msgtype::regular, payload_size,
          (size_t&)s_mem[s_idx::offset]);

      regular_msg* msg = new ((char*)env + sizeof(envelope)) regular_msg(
          chare_id, chare_idx, ep_id);

      if (payload_size > 0) {
        s_mem[s_idx::dst] = (uint64_t)((char*)msg + sizeof(regular_msg));
        s_mem[s_idx::src] = (uint64_t)buf;
      }
      s_mem[s_idx::size] = (uint64_t)payload_size;
      s_mem[s_idx::local_rank] = (uint64_t)get_local_rank_from_pe(dst_pe);
    } else {
      // Need to send to another device
      // Create and send request to CE
      env = create_envelope(msgtype::request, 0, (size_t&)s_mem[s_idx::offset]);

      request_msg* msg = new ((char*)env + sizeof(envelope)) request_msg(
          chare_id, chare_idx, ep_id, msgtype::regular, buf, payload_size, dst_pe);
      s_mem[s_idx::size] = 0;
      int my_ce = get_ce_from_pe((int)s_mem[s_idx::my_pe]);
      s_mem[s_idx::local_rank] = (uint64_t)get_local_rank_from_ce(my_ce);
    }
    s_mem[s_idx::env] = (uint64_t)env;
  }
  __syncthreads();

  // Fill in payload (from regular GPU memory to NVSHMEM symmetric memory)
  if (s_mem[s_idx::size] > 0) {
    memcpy_kernel((void*)s_mem[s_idx::dst], (void*)s_mem[s_idx::src],
        (size_t)s_mem[s_idx::size]);
  }

  // Send a local message either directly to dst PE or to a responsible CE
  send_local_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
      (int)s_mem[s_idx::local_rank]);
}

__device__ void charm::send_delegate_msg(request_msg* req) {
  // Prepare message for sending remotely
  if (threadIdx.x == 0) {
    envelope* env = nullptr;
    if (req->type == msgtype::regular) {
      // Need to send to CE responsible for target PE
      env = create_envelope(msgtype::forward, req->payload_size,
          (size_t&)s_mem[s_idx::offset]);

      forward_msg* msg = new ((char*)env + sizeof(envelope)) forward_msg(
          req->chare_id, req->chare_idx, req->ep_id, req->dst_pe);

      if (req->payload_size > 0) {
        s_mem[s_idx::dst] = (uint64_t)((char*)msg + sizeof(forward_msg));
        s_mem[s_idx::src] = (uint64_t)req->buf;
        s_mem[s_idx::size] = (uint64_t)req->payload_size;
      }
      s_mem[s_idx::dst_ce] = (uint64_t)get_ce_from_pe(req->dst_pe);
    } else if (req->type == msgtype::begin_terminate) {
      // Send begin termination message to CE 0
      env = create_envelope(msgtype::begin_terminate, 0,
          (size_t&)s_mem[s_idx::offset]);

      s_mem[s_idx::dst_ce] = 0;
    } else {
      PERROR("CE %d invalid message type %d in send_delegate_msg\n",
          (int)s_mem[s_idx::my_ce], req->type);
      assert(false);
    }
    s_mem[s_idx::env] = (uint64_t)env;
  }
  __syncthreads();

  // Fill in payload
  if (req->payload_size > 0) {
    memcpy_kernel((void*)s_mem[s_idx::dst], (void*)s_mem[s_idx::src],
        (size_t)s_mem[s_idx::size]);
  }

  send_remote_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
      (int)s_mem[s_idx::dst_ce]);
}

// TODO
__device__ __forceinline__ void send_user_msg_common(int chare_id, int chare_idx,
    int ep_id, const message& msg) {
  envelope* env = msg.env;
  if (threadIdx.x == 0) {
    // Set regular message fields using placement new
    new ((char*)env + sizeof(envelope)) regular_msg(chare_id, chare_idx, ep_id);
  }
  __syncthreads();

  //send_msg(env, msg.offset, msg.dst_pe);
}

__device__ void charm::send_user_msg(int chare_id, int chare_idx, int ep_id,
    const message& msg) {
  send_user_msg_common(chare_id, chare_idx, ep_id, msg);
}

__device__ void charm::send_user_msg(int chare_id, int chare_idx, int ep_id,
    const message& msg, size_t payload_size) {
  // Send size can be smaller than allocated message size
  msg.env->size = envelope::alloc_size(msgtype::user, payload_size);

  send_user_msg_common(chare_id, chare_idx, ep_id, msg);
}

__device__ void charm::send_begin_term_msg() {
  comm* c = (comm*)(s_mem + SMEM_CNT_MAX);

  // Don't do anything if message to begin termination
  // has already been sent from this PE
  if (c->sent_term_flag) return;

  if (threadIdx.x == 0) {
    c->sent_term_flag = true;

    int src_pe = s_mem[s_idx::my_pe];
    int src_dev = get_dev_from_pe(src_pe);
    int dst_dev = get_dev_from_ce(0);
    envelope* env = nullptr;
    if (src_dev == dst_dev) {
      // CE 0 is on the same device, send directly
      env = create_envelope(msgtype::begin_terminate, 0,
          (size_t&)s_mem[s_idx::offset]);
      s_mem[s_idx::local_rank] = (uint64_t)get_local_rank_from_ce(0);
    } else {
      // CE 0 is on a difference device, delegate to CE
      env = create_envelope(msgtype::request, 0,
          (size_t&)s_mem[s_idx::offset]);

      request_msg* msg = new ((char*)env + sizeof(envelope)) request_msg(
          -1, -1, -1, msgtype::begin_terminate, nullptr, 0, -1);
      int src_ce = get_ce_from_pe(src_pe);
      s_mem[s_idx::local_rank] = (uint64_t)get_local_rank_from_ce(src_ce);
    }
    s_mem[s_idx::env] = (uint64_t)env;
  }
  __syncthreads();

  // Send message to CE 0 (either directly or indirectly)
  send_local_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
      (int)s_mem[s_idx::local_rank]);
}

__device__ void charm::send_do_term_msg_ce(int dst_ce) {
  // Prepare message
  if (threadIdx.x == 0) {
    envelope* env = create_envelope(msgtype::do_terminate, 0,
        (size_t&)s_mem[s_idx::offset]);
    s_mem[s_idx::env] = (uint64_t)env;
  }
  __syncthreads();

  // Send message (dst CE could be on the same device or remote)
  int src_ce = s_mem[s_idx::my_ce];
  int src_dev = get_dev_from_ce(src_ce);
  int dst_dev = get_dev_from_ce(dst_ce);
  if (src_dev == dst_dev) {
    send_local_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
        get_local_rank_from_ce(dst_ce));
  } else {
    send_remote_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
        dst_ce);
  }
}

__device__ void charm::send_do_term_msg_pe(int dst_local_rank) {
  // Prepare message
  if (threadIdx.x == 0) {
    envelope* env = create_envelope(msgtype::do_terminate, 0,
        (size_t&)s_mem[s_idx::offset]);
    s_mem[s_idx::env] = (uint64_t)env;
  }
  __syncthreads();

  // Send message to child PE
  send_local_msg((envelope*)s_mem[s_idx::env], (size_t)s_mem[s_idx::offset],
      dst_local_rank);
}
