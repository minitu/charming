#include "hip/hip_runtime.h"
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include "charming.h"
#include "message.h"
#include "comm.h"
#include "scheduler.h"
#include "chare.h"
#include "util.h"

using namespace charm;

// GPU constant memory
extern __constant__ int c_n_sms;
extern __constant__ int c_my_dev;
extern __constant__ int c_my_dev_node;
extern __constant__ int c_n_devs;
extern __constant__ int c_n_devs_node;
extern __constant__ int c_n_pes;
extern __constant__ int c_n_pes_node;
extern __constant__ int c_n_nodes;

// GPU global memory
extern __device__ chare_proxy_base* chare_proxies[];

// GPU shared memory
extern __shared__ uint64_t s_mem[];

__device__ msgtype charm::process_msg(void* addr, ssize_t* processed_size,
    bool& begin_term_flag, bool& do_term_flag) {
  int my_pe = s_mem[3];
  envelope* env = (envelope*)addr;
  msgtype type = env->type;
  if (threadIdx.x == 0) {
    if (processed_size) *processed_size = env->size;
    PDEBUG("PE %d: received msg type %d size %llu from PE %d\n",
           my_pe, type, env->size, env->src_pe);
  }
  __syncthreads();

  if (type == msgtype::create) {
    // Creation message
    create_msg* msg = (create_msg*)((char*)env + sizeof(envelope));
    if (threadIdx.x == 0) {
      PDEBUG("PE %d: creation msg chare ID %d, n_local %d, n_total %d, "
          "start idx %d, end idx %d\n", my_pe, msg->chare_id, msg->n_local,
          msg->n_total, msg->start_idx, msg->end_idx);
    }

    chare_proxy_base*& chare_proxy = chare_proxies[msg->chare_id];
    char* map_ptr = (char*)msg + sizeof(create_msg);
    char* obj_ptr = map_ptr + sizeof(int) * msg->n_total;

    chare_proxy->create_local(msg->n_local, msg->n_total, msg->start_idx,
        msg->end_idx, map_ptr, obj_ptr);

  } else if (type == msgtype::regular || type == msgtype::user) {
    // Regular message (including user message)
    regular_msg* msg = (regular_msg*)((char*)env + sizeof(envelope));
    if (threadIdx.x == 0) {
      PDEBUG("PE %d: regular msg chare ID %d chare idx %d EP ID %d\n", my_pe,
          msg->chare_id, msg->chare_idx, msg->ep_id);
    }

    chare_proxy_base*& chare_proxy = chare_proxies[msg->chare_id];
    void* payload = (char*)msg + sizeof(regular_msg);

    chare_proxy->call(msg->chare_idx, msg->ep_id, payload);

  } else if (type == msgtype::begin_terminate) {
    // Should only be received by PE 0
    assert(my_pe == 0);

    // Begin termination message
    if (threadIdx.x == 0) {
    PDEBUG("PE %d: begin terminate msg\n", my_pe);
      if (!begin_term_flag) {
        for (int i = 0; i < n_pes(); i++) {
          send_do_term_msg(i);
        }
        begin_term_flag = true;
      }
    }
    __syncthreads();

  } else if (type == msgtype::do_terminate) {
    // Do termination message
    if (threadIdx.x == 0) {
      PDEBUG("PE %d: do terminate msg\n", my_pe);
      do_term_flag = true;
    }
    __syncthreads();

  } else {
    if (threadIdx.x == 0) {
      PERROR("PE %d: unrecognized message type %d\n", my_pe, type);
    }
    assert(false);
  }

  return type;
}

__device__ __forceinline__ void loop(comm* c) {
  c->process_local();
  c->process_remote();
  c->cleanup();
}

__global__ void charm::scheduler(int argc, char** argv, size_t* argvs) {
  // For inter-PE/TB synchronization
  cg::grid_group grid = cg::this_grid();

  // Register user chares and entry methods
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  if (gid == 0) {
    chare_proxy_cnt = 0;
    register_chares();
  }

  // Communication module resides in shared memory (one per PE/TB)
  comm* c = (comm*)(s_mem+SMEM_CNT_MAX);
  if (threadIdx.x == 0) {
    // Initialize comm module
    c->init();

    // Store my PE number in shared memory
    s_mem[3] = c_my_dev * c_n_sms + blockIdx.x;
    s_mem[4] = c_my_dev_node * c_n_sms + blockIdx.x;
  }
  __syncthreads();

  // Global synchronization
  if (gid == 0) {
    nvshmem_barrier_all();
  }
  grid.sync();

  int my_pe = s_mem[3];
  if (my_pe == 0) {
    // Execute user's main function
    main(argc, argv, argvs);
  }

  // Global synchronization
  if (gid == 0) {
    nvshmem_barrier_all();
  }
  grid.sync();

  // Loop until termination
  do {
    loop(c);
  } while (!c->do_term_flag);

  if (threadIdx.x == 0) {
    PDEBUG("PE %d terminating...\n", my_pe);
  }
}
