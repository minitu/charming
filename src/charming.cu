#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <string.h>
#include <hip/hip_runtime.h>
#ifdef CHARMING_USE_MPI
#include <mpi.h>
#endif
#include <nvshmem.h>
#include <nvshmemx.h>

#include "charming.h"
#include "common.h"
#include "message.h"
#include "comm.h"
#include "scheduler.h"
#include "util.h"

#define NVSHMEM_MAX_SIZE 2147483648

using namespace charm;

hipStream_t stream;

// GPU constant memory
__constant__ int c_n_sms;
__constant__ int c_n_clusters;;
__constant__ int c_n_workers;
__constant__ int c_my_dev;
__constant__ int c_my_dev_node;
__constant__ int c_n_devs;
__constant__ int c_n_devs_node;
__constant__ int c_n_nodes;
__constant__ int c_n_pes;
__constant__ int c_n_pes_node;

// GPU global memory
__device__ __managed__ chare_proxy_table* proxy_tables;

// GPU shared memory
extern __shared__ uint64_t s_mem[];

int main(int argc, char* argv[]) {
  // Increase maximum NVSHMEM memory size
  std::string env_str = "NVSHMEM_SYMMETRIC_SIZE=";
  env_str += std::to_string(NVSHMEM_MAX_SIZE);
  putenv(const_cast<char*>(env_str.c_str()));

  // Initialize NVSHMEM (and MPI if needed)
#ifdef CHARMING_USE_MPI
  MPI_Init(&argc, &argv);
  int world_size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
#else
  nvshmem_init();
#endif

  // Execution environment
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  //int max_threads_tb = prop.maxThreadsPerBlock;
  int h_n_sms = prop.multiProcessorCount;
  //int h_n_sms = 1;
  int h_n_clusters = 2;
  int h_n_workers = (h_n_sms / h_n_clusters) - 1;
  int h_my_dev = nvshmem_my_pe();
  int h_my_dev_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  int h_n_devs = nvshmem_n_pes();
  int h_n_devs_node = nvshmem_team_n_pes(NVSHMEMX_TEAM_NODE);
  int h_n_nodes = h_n_devs / h_n_devs_node;
  int h_n_pes = h_n_devs * h_n_clusters * h_n_workers;
  int h_n_pes_node = h_n_pes / h_n_nodes;

  // Check if number of PE clusters is valid
  if (h_n_sms % h_n_clusters != 0) {
    if (h_my_dev == 0) {
      PERROR("Number of PE clusters must be a factor of the number of SMs\n");
    }
    return -1;
  }

  // Check for necessary CUDA functionalities
  if (!prop.cooperativeLaunch) {
    if (h_my_dev == 0) {
      PERROR("Need support for CUDA Cooperative Groups\n");
    }
    return -1;
  }
  if (!prop.managedMemory) {
    if (h_my_dev == 0) {
      PERROR("Need support for CUDA Unified Memory\n");
    }
    return -1;
  }

  // Initialize CUDA and create stream
  // Round-robin mapping of processes to GPUs
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    if (h_my_dev == 0) {
      PERROR("Need at least 1 GPU but detected %d GPUs\n", n_devices);
    }
    return -1;
  }
  hipSetDevice(h_my_dev_node % n_devices);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  cuda_check_error();

  // Create chare proxy tables
  hipMallocManaged(&proxy_tables, sizeof(chare_proxy_table) * h_n_sms);
  for (int i = 0; i < h_n_sms; i++) {
    new (&proxy_tables[i]) chare_proxy_table();
  }

  // Transfer command line arguments to GPU
  char* m_args; // Contains the actual arguments consecutively
  char** m_argv; // Contains pointers to the arguments
  size_t* m_argvs; // Contains sizes of the arguments
  size_t argvs_total = 0; // Sum of all argument sizes

  // Figure out size of each argument and total size
  hipMallocManaged(&m_argvs, sizeof(size_t) * argc);
  for (int i = 0; i < argc; i++) {
    m_argvs[i] = strlen(argv[i]);
    argvs_total += m_argvs[i] + 1; // Include NULL character
  }

  // Allocate memory for actual arguments
  hipMallocManaged(&m_args, argvs_total);

  // Copy arguments into managed memory and store their addresses
  hipMallocManaged(&m_argv, sizeof(char*) * argc);
  char* cur_arg = m_args;
  for (int i = 0; i < argc; i++) {
    strcpy(cur_arg, argv[i]);
    m_argv[i] = cur_arg;

    cur_arg += m_argvs[i] + 1; // Include NULL character
  }

  // Transfer execution environment constants
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_sms), &h_n_sms, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_clusters), &h_n_clusters, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_workers), &h_n_workers, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_dev), &h_my_dev, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_dev_node), &h_my_dev_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_devs), &h_n_devs, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_devs_node), &h_n_devs_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes), &h_n_pes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes_node), &h_n_pes_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_nodes), &h_n_nodes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  cuda_check_error();

  // Change device limits
  size_t stack_size, heap_size;
  size_t smem_size = SMEM_CNT_MAX * sizeof(uint64_t) + 128;
  constexpr size_t new_stack_size = 16384;
  hipDeviceSetLimit(hipLimitStackSize, new_stack_size);
  hipDeviceGetLimit(&stack_size, hipLimitStackSize);
  constexpr size_t new_heap_size = 1073741824;
  hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size);
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
  hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  cuda_check_error();

  // Print configuration
  dim3 grid_dim = dim3(h_n_sms);
  //dim3 block_dim = dim3(max_threads_tb);
  dim3 block_dim = dim3(512);
  int max_blocks_sm;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks_sm, (const void*)scheduler,
      block_dim.x*block_dim.y*block_dim.z, 0);
  cuda_check_error();
  if (h_my_dev == 0) {
    PINFO("Initiating CharminG\n");
    PINFO("PEs: %d, GPU Devices: %d, Nodes: %d\n", h_n_pes, h_n_devs, h_n_nodes);
    PINFO("PE Clusters: %d, Workers: %d\n", h_n_clusters, h_n_workers);
    PINFO("Thread grid: %d x %d x %d, Thread block: %d x %d x %d\n",
        grid_dim.x, grid_dim.y, grid_dim.z, block_dim.x, block_dim.y, block_dim.z);
    PINFO("Stack size: %llu Bytes, Heap size: %llu Bytes\n", stack_size, heap_size);
    PINFO("Shared memory size: %llu Bytes, Clock rate: %.2lf GHz\n",
        smem_size, (double)prop.clockRate / 1e6);
    PINFO("Max active TBs per SM: %d, Number of SMs: %d\n", max_blocks_sm, h_n_sms);
  }

  // Initialize communication module
  comm_init_host(h_n_pes, h_n_sms, h_n_clusters);
  nvshmemx_barrier_all_on_stream(stream);

  // Launch scheduler kernel
  void* kargs[] = { &argc, &m_argv, &m_argvs };
  nvshmemx_collective_launch((const void*)scheduler, grid_dim, block_dim, kargs,
      smem_size, stream);
  hipStreamSynchronize(stream);
  cuda_check_error();
  nvshmemx_barrier_all_on_stream(stream);

  if (h_my_dev == 0) {
    PINFO("Exiting CharminG\n");
  }

  // Cleanup
  comm_fini_host();
  hipFree(proxy_tables);
  hipStreamDestroy(stream);
  nvshmem_finalize();
#ifdef CHARMING_USE_MPI
  MPI_Finalize();
#endif

  return 0;
}

__device__ void charm::end() {
  // TODO: Check if begin_terminate message has already been sent from this PE
  send_term_msg(true, 0);
}

__device__ void charm::abort() {
  // Abort currently running PE
  if (threadIdx.x == 0) {
    comm* c = (comm*)(s_mem + SMEM_CNT_MAX);
    c->do_term_flag = true;
  }
  __syncthreads();
}

__device__ int charm::my_pe() { return s_mem[s_idx::my_pe]; }
__device__ int charm::n_pes() { return c_n_pes; }
__device__ int charm::my_pe_node() { return s_mem[s_idx::my_pe_node]; }
__device__ int charm::n_pes_node() { return c_n_pes_node; }
__device__ int charm::n_nodes() { return c_n_nodes; }

__device__ int charm::device_atoi(const char* str, int strlen) {
  int tmp = 0;
  for (int i = 0; i < strlen; i++) {
    int multiplier = 1;
    for (int j = 0; j < strlen - i - 1; j++) {
      multiplier *= 10;
    }
    tmp += (str[i] - 48) * multiplier;
  }
  return tmp;
}
