#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#include "charming.h"
#include "message.h"
#include "scheduler.h"
#include "ringbuf.h"
#include "util.h"

// Maximum number of chare types
#define CHARE_TYPE_CNT_MAX 1024

using namespace charm;

__constant__ int c_my_pe;
__constant__ int c_n_pes;

__device__ spsc_ringbuf_t* mbuf;
__device__ size_t mbuf_size;
__device__ uint64_t* used_arr;
__device__ uint64_t* addr_arr;
__device__ uint64_t* size_arr;
__device__ size_t arr_size;

__device__ chare_proxy_base* chare_proxies[CHARE_TYPE_CNT_MAX];
__device__ int chare_proxy_cnt;

int main(int argc, char* argv[]) {
  // Initialize MPI
  MPI_Init(&argc, &argv);
  int world_size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Initialize NVSHMEM
  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
  int h_my_pe = nvshmem_my_pe();
  int h_n_pes = nvshmem_n_pes();

  // Initialize CUDA
  // FIXME: Always mapped to first device
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    if (rank == 0) {
      printf("ERROR: Need at least 1 GPU but detected %d GPUs\n", n_devices);
    }
    return -1;
  }
  hipSetDevice(0);
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  cuda_check_error();

  // Transfer command line arguments to GPU
  size_t h_argvs[argc];
  size_t argvs_total = 0;
  for (int i = 0; i < argc; i++) {
    h_argvs[i] = strlen(argv[i]);
    argvs_total += h_argvs[i] + 1; // Include NULL character
  }
  size_t* d_argvs;
  hipMalloc(&d_argvs, sizeof(size_t) * argc);
  hipMemcpyAsync(d_argvs, h_argvs, sizeof(size_t) * argc, hipMemcpyHostToDevice, stream);
  char* d_argvv;
  hipMalloc(&d_argvv, argvs_total);
  char* h_argv[argc];
  h_argv[0] = d_argvv;
  hipMemcpyAsync(h_argv[0], argv[0], h_argvs[0] + 1, hipMemcpyHostToDevice, stream);
  for (int i = 1; i < argc; i++) {
    h_argv[i] = h_argv[i-1] + h_argvs[i-1] + 1;
    hipMemcpyAsync(h_argv[i], argv[i], h_argvs[i] + 1, hipMemcpyHostToDevice, stream);
  }
  char** d_argv;
  hipMalloc(&d_argv, sizeof(char*) * argc);
  hipMemcpyAsync(d_argv, h_argv, sizeof(char*) * argc, hipMemcpyHostToDevice, stream);
  cuda_check_error();

  // Allocate message buffer and signal arrays using NVSHMEM
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  //size_t h_mbuf_size = prop.totalGlobalMem / 2;
  size_t h_mbuf_size = 1073741824;
  spsc_ringbuf_t* h_mbuf = spsc_ringbuf_malloc(h_mbuf_size);
  size_t h_arr_size = MSG_IN_FLIGHT_MAX * h_n_pes * sizeof(uint64_t);
  uint64_t* h_used_arr = (uint64_t*)nvshmem_malloc(h_arr_size);
  uint64_t* h_addr_arr = (uint64_t*)nvshmem_malloc(h_arr_size);
  uint64_t* h_size_arr = (uint64_t*)nvshmem_malloc(h_arr_size);
  assert(h_used_arr && h_addr_arr && h_size_arr);
  cuda_check_error();

  // Synchronize all NVSHMEM PEs
  nvshmem_barrier_all();

  // Change device limits
  size_t stack_size, heap_size;
  //size_t new_heap_size = 8589934592; // Set max heap size to 8GB
  //hipDeviceSetLimit(hipLimitStackSize, 16384);
  hipDeviceGetLimit(&stack_size, hipLimitStackSize);
  //hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size);
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

  // Print configuration and launch scheduler
  /*
  int grid_size = prop.multiProcessorCount;
  int block_size = prop.maxThreadsPerBlock;
  */
  int grid_size = 1;
  int block_size = 1;
  if (rank == 0) {
    printf("CHARMING\nGrid size: %d\nBlock size: %d\nStack size: %llu B\n"
           "Heap size: %llu B\nClock rate: %.2lf GHz\n",
           grid_size, block_size, stack_size, heap_size,
           (double)prop.clockRate / 1e6);
  }
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_pe), &h_my_pe, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes), &h_n_pes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(mbuf), &h_mbuf, sizeof(spsc_ringbuf_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(mbuf_size), &h_mbuf_size, sizeof(size_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(used_arr), &h_used_arr, sizeof(uint64_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(addr_arr), &h_addr_arr, sizeof(uint64_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(size_arr), &h_size_arr, sizeof(uint64_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(arr_size), &h_arr_size, sizeof(size_t), 0, hipMemcpyHostToDevice, stream);

  cuda_check_error();
  /* This doesn't support CUDA dynamic parallelism, will it be a problem?
  void* scheduler_args[4] = { &rbuf, &rbuf_size, &mbuf, &mbuf_size };
  nvshmemx_collective_launch((const void*)scheduler, grid_size, block_size,
      //scheduler_args, 0, stream);
      nullptr, 0, stream);
  */
  scheduler<<<grid_size, block_size, 0, stream>>>(argc, d_argv, d_argvs);
  hipStreamSynchronize(stream);
  cuda_check_error();

  //nvshmemx_barrier_all_on_stream(stream); // Hangs
  nvshmem_barrier_all();

  // Cleanup
  nvshmem_free(h_used_arr);
  nvshmem_free(h_addr_arr);
  nvshmem_free(h_size_arr);
  spsc_ringbuf_free(h_mbuf);
  hipStreamDestroy(stream);
  nvshmem_finalize();
  MPI_Finalize();

  return 0;
}

__device__ void charm::end() {
  // TODO: Check if begin_terminate message has already been sent from this PE
  send_begin_term_msg(0);
}

__device__ int charm::n_pes() {
  return c_n_pes;
}

__device__ int charm::my_pe() {
  return c_my_pe;
}

__device__ int charm::device_atoi(const char* str, int strlen) {
  int tmp = 0;
  for (int i = 0; i < strlen; i++) {
    int multiplier = 1;
    for (int j = 0; j < strlen - i - 1; j++) {
      multiplier *= 10;
    }
    tmp += (str[i] - 48) * multiplier;
  }
  return tmp;
}
