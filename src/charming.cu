#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#ifdef CHARMING_USE_MPI
#include <mpi.h>
#endif
#include <nvshmem.h>
#include <nvshmemx.h>

#include "charming.h"
#include "message.h"
#include "comm.h"
#include "scheduler.h"
#include "util.h"

// Maximum number of chare types
#define CHARE_TYPE_CNT_MAX 1024

using namespace charm;

hipStream_t stream;

__constant__ int c_my_pe;
__constant__ int c_n_pes;
__constant__ int c_my_pe_node;
__constant__ int c_n_pes_node;
__constant__ int c_n_nodes;

__device__ chare_proxy_base* chare_proxies[CHARE_TYPE_CNT_MAX];
__device__ int chare_proxy_cnt;

int main(int argc, char* argv[]) {
#ifdef CHARMING_USE_MPI
  // Initialize MPI
  MPI_Init(&argc, &argv);
  int world_size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Initialize NVSHMEM
  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
#else
  nvshmem_init();
#endif // CHARMING_USE_MPI
  int h_my_pe = nvshmem_my_pe();
  int h_n_pes = nvshmem_n_pes();
  int h_my_pe_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  int h_n_pes_node = nvshmem_team_n_pes(NVSHMEMX_TEAM_NODE);
  int h_n_nodes = h_n_pes / h_n_pes_node;

  // Initialize CUDA and create stream
  // Round-robin mapping of processes to GPUs
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    if (h_my_pe == 0) {
      PERROR("Need at least 1 GPU but detected %d GPUs\n", n_devices);
    }
    return -1;
  }
  hipSetDevice(h_my_pe_node % n_devices);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  cuda_check_error();

  // Transfer command line arguments to GPU
  size_t h_argvs[argc];
  size_t argvs_total = 0;
  for (int i = 0; i < argc; i++) {
    h_argvs[i] = strlen(argv[i]);
    argvs_total += h_argvs[i] + 1; // Include NULL character
  }
  size_t* d_argvs;
  hipMalloc(&d_argvs, sizeof(size_t) * argc);
  hipMemcpyAsync(d_argvs, h_argvs, sizeof(size_t) * argc, hipMemcpyHostToDevice, stream);
  char* d_argvv;
  hipMalloc(&d_argvv, argvs_total);
  char* h_argv[argc];
  h_argv[0] = d_argvv;
  hipMemcpyAsync(h_argv[0], argv[0], h_argvs[0] + 1, hipMemcpyHostToDevice, stream);
  for (int i = 1; i < argc; i++) {
    h_argv[i] = h_argv[i-1] + h_argvs[i-1] + 1;
    hipMemcpyAsync(h_argv[i], argv[i], h_argvs[i] + 1, hipMemcpyHostToDevice, stream);
  }
  char** d_argv;
  hipMalloc(&d_argv, sizeof(char*) * argc);
  hipMemcpyAsync(d_argv, h_argv, sizeof(char*) * argc, hipMemcpyHostToDevice, stream);

  // Transfer constants
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_pe), &h_my_pe, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes), &h_n_pes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_pe_node), &h_my_pe_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes_node), &h_n_pes_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_nodes), &h_n_nodes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  cuda_check_error();

  // Change device limits
  size_t stack_size, heap_size;
  constexpr size_t new_stack_size = 16384;
  hipDeviceSetLimit(hipLimitStackSize, new_stack_size);
  hipDeviceGetLimit(&stack_size, hipLimitStackSize);
  //constexpr size_t new_heap_size = 8589934592; // Set max heap size to 8GB
  //hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size);
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

  // Print configuration and launch scheduler
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int max_blocks_sm;
  int n_sms = prop.multiProcessorCount;
  int max_threads_tb = prop.maxThreadsPerBlock;
  dim3 grid_dim = dim3(n_sms);
  dim3 block_dim = dim3(max_threads_tb);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks_sm, (const void*)scheduler,
      block_dim.x*block_dim.y*block_dim.z, 0);
  cuda_check_error();
  if (h_my_pe == 0) {
    PINFO("Initiating CharminG\n");
    PINFO("PEs: %d, Nodes: %d\n", h_n_pes, h_n_nodes);
    PINFO("Thread grid: %d x %d x %d, Thread block: %d x %d x %d\n",
        grid_dim.x, grid_dim.y, grid_dim.z, block_dim.x, block_dim.y, block_dim.z);
    PINFO("Stack size: %llu Bytes, Heap size: %llu Bytes, Clock rate: %.2lf GHz\n",
        stack_size, heap_size, (double)prop.clockRate / 1e6);
    PINFO("Max active TBs per SM: %d, Number of SMs: %d\n", max_blocks_sm, n_sms);
  }

  // Initialize communication module
  comm_init_host(h_n_pes);
  nvshmemx_barrier_all_on_stream(stream);

  // Launch scheduler kernel
  void* kargs[] = { &argc, &d_argv, &d_argvs };
  nvshmemx_collective_launch((const void*)scheduler, grid_dim, block_dim, kargs, 0, stream);
  hipStreamSynchronize(stream);
  cuda_check_error();
  nvshmemx_barrier_all_on_stream(stream);

  // Cleanup
  comm_fini_host(h_n_pes);
  hipStreamDestroy(stream);
  nvshmem_finalize();
#ifdef CHARMING_USE_MPI
  MPI_Finalize();
#endif

  return 0;
}

__device__ void charm::end() {
  // TODO: Check if begin_terminate message has already been sent from this PE
  send_begin_term_msg(0);
}

__device__ int charm::my_pe() { return c_my_pe; }
__device__ int charm::n_pes() { return c_n_pes; }
__device__ int charm::my_pe_node() { return c_my_pe_node; }
__device__ int charm::n_pes_node() { return c_n_pes_node; }
__device__ int charm::n_nodes() { return c_n_nodes; }

__device__ int charm::device_atoi(const char* str, int strlen) {
  int tmp = 0;
  for (int i = 0; i < strlen; i++) {
    int multiplier = 1;
    for (int j = 0; j < strlen - i - 1; j++) {
      multiplier *= 10;
    }
    tmp += (str[i] - 48) * multiplier;
  }
  return tmp;
}
