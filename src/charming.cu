#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include "charming.h"
#include "message.h"
#include "scheduler.h"
#include "ringbuf.h"
#include "util.h"

#define CHARE_TYPE_CNT_MAX 1024 // Maximum number of chare types

__device__ mpsc_ringbuf_t* rbuf;
__device__ size_t rbuf_size;
__device__ spsc_ringbuf_t* mbuf;
__device__ size_t mbuf_size;

using namespace charm;

__device__ chare_proxy_base* chare_proxies[CHARE_TYPE_CNT_MAX];
__device__ int chare_proxy_cnt;

int main(int argc, char* argv[]) {
  int rank;
  hipStream_t stream;

  // Initialize MPI
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Initialize NVSHMEM
  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  // Initialize CUDA
  hipSetDevice(0);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // Transfer command line arguments to GPU
  size_t h_argvs[argc];
  size_t argvs_total = 0;
  for (int i = 0; i < argc; i++) {
    h_argvs[i] = strlen(argv[i]);
    argvs_total += h_argvs[i] + 1; // Include NULL character
  }
  size_t* d_argvs;
  hipMalloc(&d_argvs, sizeof(size_t) * argc);
  hipMemcpyAsync(d_argvs, h_argvs, sizeof(size_t) * argc, hipMemcpyHostToDevice, stream);
  char* d_argvv;
  hipMalloc(&d_argvv, argvs_total);
  char* h_argv[argc];
  h_argv[0] = d_argvv;
  hipMemcpyAsync(h_argv[0], argv[0], h_argvs[0] + 1, hipMemcpyHostToDevice, stream);
  for (int i = 1; i < argc; i++) {
    h_argv[i] = h_argv[i-1] + h_argvs[i-1] + 1;
    hipMemcpyAsync(h_argv[i], argv[i], h_argvs[i] + 1, hipMemcpyHostToDevice, stream);
  }
  char** d_argv;
  hipMalloc(&d_argv, sizeof(char*) * argc);
  hipMemcpyAsync(d_argv, h_argv, sizeof(char*) * argc, hipMemcpyHostToDevice, stream);

  // Allocate message queue with NVSHMEM
  size_t h_rbuf_size = (1 << 28);
  mpsc_ringbuf_t* h_rbuf = mpsc_ringbuf_malloc(h_rbuf_size);
  size_t h_mbuf_size = (1 << 28);
  spsc_ringbuf_t* h_mbuf = spsc_ringbuf_malloc(h_mbuf_size);
  nvshmem_barrier_all();

  // Launch scheduler
  int grid_size = 1;
  int block_size = 1;
  //hipDeviceSetLimit(hipLimitStackSize, 16384);
  size_t stack_size;
  hipDeviceGetLimit(&stack_size, hipLimitStackSize);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  if (!rank) {
    printf("CHARMING\nGrid size: %d\nBlock size: %d\nStack size: %llu\nClock rate: %.2lf GHz\n",
           grid_size, block_size, stack_size, (double)prop.clockRate / 1e6);
  }
  //void* scheduler_args[4] = { &rbuf, &rbuf_size, &mbuf, &mbuf_size };
  hipMemcpyToSymbolAsync(HIP_SYMBOL(rbuf), &h_rbuf, sizeof(mpsc_ringbuf_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(rbuf_size), &h_rbuf_size, sizeof(size_t), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(mbuf), &h_mbuf, sizeof(spsc_ringbuf_t*), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(mbuf_size), &h_mbuf_size, sizeof(size_t), 0, hipMemcpyHostToDevice, stream);
  /* TODO: This doesn't support CUDA dynamic parallelism, will it be a problem?
  nvshmemx_collective_launch((const void*)scheduler, grid_size, block_size,
      //scheduler_args, 0, stream);
      nullptr, 0, stream);
      */
  scheduler<<<grid_size, block_size, 0, stream>>>(argc, d_argv, d_argvs);
  cuda_check_error();
  hipStreamSynchronize(stream);
  //nvshmemx_barrier_all_on_stream(stream); // Hangs
  nvshmem_barrier_all();

  // Finalize NVSHMEM and MPI
  spsc_ringbuf_free(h_mbuf);
  mpsc_ringbuf_free(h_rbuf);
  nvshmem_finalize();
  hipStreamDestroy(stream);
  MPI_Finalize();

  return 0;
}

__device__ void charm::exit() {
  int n_pes = nvshmem_n_pes();
  for (int pe = 0; pe < n_pes; pe++) {
    send_term_msg(pe);
  }
}
