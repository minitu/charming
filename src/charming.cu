#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#ifdef CHARMING_USE_MPI
#include <mpi.h>
#endif
#include <nvshmem.h>
#include <nvshmemx.h>

#include "charming.h"
#include "message.h"
#include "comm.h"
#include "scheduler.h"
#include "util.h"

// Maximum number of chare types
#define CHARE_TYPE_CNT_MAX 1024

using namespace charm;

hipStream_t stream;

// GPU constant memory
__constant__ int c_n_sms;
__constant__ int c_my_dev;
__constant__ int c_my_dev_node;
__constant__ int c_n_devs;
__constant__ int c_n_devs_node;
__constant__ int c_n_pes;
__constant__ int c_n_pes_node;
__constant__ int c_n_nodes;

// GPU global memory
__device__ chare_proxy_base* chare_proxies[CHARE_TYPE_CNT_MAX];
__device__ int chare_proxy_cnt;

// GPU shared memory
extern __shared__ uint64_t s_mem[];

int main(int argc, char* argv[]) {
  // Initialize NVSHMEM (and MPI if needed)
#ifdef CHARMING_USE_MPI
  MPI_Init(&argc, &argv);
  int world_size;
  int rank;
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
#else
  nvshmem_init();
#endif

  // Execution environment
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int max_threads_tb = prop.maxThreadsPerBlock;
  int h_n_sms = prop.multiProcessorCount;
  int h_my_dev = nvshmem_my_pe();
  int h_my_dev_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  int h_n_devs = nvshmem_n_pes();
  int h_n_devs_node = nvshmem_team_n_pes(NVSHMEMX_TEAM_NODE);
  int h_n_pes = h_n_devs * h_n_sms;
  int h_n_pes_node = h_n_devs_node * h_n_sms;
  int h_n_nodes = h_n_devs / h_n_devs_node;

  // Initialize CUDA and create stream
  // Round-robin mapping of processes to GPUs
  int n_devices = 0;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    if (h_my_dev == 0) {
      PERROR("Need at least 1 GPU but detected %d GPUs\n", n_devices);
    }
    return -1;
  }
  hipSetDevice(h_my_dev_node % n_devices);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  cuda_check_error();

  // Transfer command line arguments to GPU
  char* m_args; // Contains the actual arguments consecutively
  char** m_argv; // Contains pointers to the arguments
  size_t* m_argvs; // Contains sizes of the arguments
  size_t argvs_total = 0; // Sum of all argument sizes

  // Figure out size of each argument and total size
  hipMallocManaged(&m_argvs, sizeof(size_t) * argc);
  for (int i = 0; i < argc; i++) {
    m_argvs[i] = strlen(argv[i]);
    argvs_total += m_argvs[i] + 1; // Include NULL character
  }

  // Allocate memory for actual arguments
  hipMallocManaged(&m_args, argvs_total);

  // Copy arguments into managed memory and store their addresses
  hipMallocManaged(&m_argv, sizeof(char*) * argc);
  char* cur_arg = m_args;
  for (int i = 0; i < argc; i++) {
    strcpy(cur_arg, argv[i]);
    m_argv[i] = cur_arg;

    cur_arg += m_argvs[i] + 1; // Include NULL character
  }

  // Transfer execution environment constants
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_sms), &h_n_sms, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_dev), &h_my_dev, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_my_dev_node), &h_my_dev_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_devs), &h_n_devs, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_devs_node), &h_n_devs_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes), &h_n_pes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_pes_node), &h_n_pes_node, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_n_nodes), &h_n_nodes, sizeof(int), 0, hipMemcpyHostToDevice, stream);
  cuda_check_error();

  // Change device limits
  size_t stack_size, heap_size;
  size_t smem_size = SMEM_CNT_MAX * sizeof(uint64_t) + 128;
  constexpr size_t new_stack_size = 16384;
  hipDeviceSetLimit(hipLimitStackSize, new_stack_size);
  hipDeviceGetLimit(&stack_size, hipLimitStackSize);
  //constexpr size_t new_heap_size = 8589934592; // Set max heap size to 8GB
  //hipDeviceSetLimit(hipLimitMallocHeapSize, new_heap_size);
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

  // Print configuration and launch scheduler
  dim3 grid_dim = dim3(h_n_sms);
  //dim3 block_dim = dim3(max_threads_tb);
  dim3 block_dim = dim3(512);
  int max_blocks_sm;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks_sm, (const void*)scheduler,
      block_dim.x*block_dim.y*block_dim.z, 0);
  cuda_check_error();
  if (h_my_dev == 0) {
    PINFO("Initiating CharminG\n");
    PINFO("PEs: %d, Nodes: %d\n", h_n_pes, h_n_nodes);
    PINFO("Thread grid: %d x %d x %d, Thread block: %d x %d x %d\n",
        grid_dim.x, grid_dim.y, grid_dim.z, block_dim.x, block_dim.y, block_dim.z);
    PINFO("Stack size: %llu Bytes, Heap size: %llu Bytes\n", stack_size, heap_size);
    PINFO("Shared memory size: %llu Bytes, Clock rate: %.2lf GHz\n",
        smem_size, (double)prop.clockRate / 1e6);
    PINFO("Max active TBs per SM: %d, Number of SMs: %d\n", max_blocks_sm, h_n_sms);
  }

  // Initialize communication module
  comm_init_host(h_n_pes, h_n_sms);
  nvshmemx_barrier_all_on_stream(stream);

  // Launch scheduler kernel
  void* kargs[] = { &argc, &m_argv, &m_argvs };
  nvshmemx_collective_launch((const void*)scheduler, grid_dim, block_dim, kargs,
      smem_size, stream);
  printf("After collective launch\n");
  hipStreamSynchronize(stream);
  cuda_check_error();
  nvshmemx_barrier_all_on_stream(stream);
  printf("After nvshmemx barrier\n");

  // Cleanup
  comm_fini_host(h_n_pes, h_n_sms);
  hipStreamDestroy(stream);
  printf("After hipStreamDestroy\n");
  nvshmem_finalize();
  printf("After nvshmemx_finalize\n");
#ifdef CHARMING_USE_MPI
  MPI_Finalize();
#endif

  return 0;
}

__device__ void charm::end() {
  // TODO: Check if begin_terminate message has already been sent from this PE
  send_begin_term_msg(0);
}

__device__ int charm::my_pe() { return s_mem[3]; }
__device__ int charm::n_pes() { return c_n_pes; }
__device__ int charm::my_pe_node() { return s_mem[4]; }
__device__ int charm::n_pes_node() { return c_n_pes_node; }
__device__ int charm::n_nodes() { return c_n_nodes; }

__device__ int charm::device_atoi(const char* str, int strlen) {
  int tmp = 0;
  for (int i = 0; i < strlen; i++) {
    int multiplier = 1;
    for (int j = 0; j < strlen - i - 1; j++) {
      multiplier *= 10;
    }
    tmp += (str[i] - 48) * multiplier;
  }
  return tmp;
}
