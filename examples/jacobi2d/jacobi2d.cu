#include "hip/hip_runtime.h"
#include <stdio.h>
#include "jacobi2d.h"

#define LEFT 0
#define RIGHT 1
#define TOP 2
#define BOTTOM 3

#define GRID_WIDTH 65536
#define GRID_HEIGHT 65536
#define N_ITERS 100
#define BLOCK_DIM 16

#define IDX(i,j) ((block_width+2)*(i) + (j))

__device__ charm::chare_proxy<Block>* block_proxy;

__device__ void charm::register_chares() {
  block_proxy = new charm::chare_proxy<Block>(2);
  block_proxy->add_entry_method(&Block::init);
  block_proxy->add_entry_method(&Block::recv_ghosts);
}

__device__ int device_atoi(const char* str, int strlen) {
  int tmp = 0;
  for (int i = 0; i < strlen; i++) {
    int multiplier = 1;
    for (int j = 0; j < strlen - i - 1; j++) {
      multiplier *= 10;
    }
    tmp += (str[i] - 48) * multiplier;
  }
  return tmp;
}

// Main
__device__ void charm::main(int argc, char** argv, size_t* argvs) {
  // Process command line arguments
  int grid_width = GRID_WIDTH;
  if (argc >= 2) grid_width = device_atoi(argv[1], argvs[1]);
  int grid_height = GRID_HEIGHT;
  if (argc >= 3) grid_height = device_atoi(argv[2], argvs[2]);
  int n_chares = charm::n_pes();
  if (argc >= 4) n_chares = device_atoi(argv[3], argvs[3]);
  int n_iters = N_ITERS;
  if (argc >= 5) n_iters = device_atoi(argv[4], argvs[4]);

  // Set up 2D grid of chares (as square as possible)
  double area[2], surf, bestsurf;
  int ipx, ipy;
  int n_chares_x, n_chares_y;
  area[0] = grid_height;
  area[1] = grid_width;
  bestsurf = 2.0 * (area[0] + area[1]);
  ipx = 1;
  while (ipx <= n_chares) {
    if (n_chares % ipx == 0) {
      ipy = n_chares / ipx;
      surf = 2.0 * (area[0] / ipx + area[1] / ipy);

      if (surf < bestsurf) {
        bestsurf = surf;
        n_chares_x = ipx;
        n_chares_y = ipy;
      }
    }
    ipx++;
  }
  if (n_chares_x * n_chares_y != n_chares) {
    printf("Decomposition failed! %d chares into %d x %d chares\n",
           n_chares, n_chares_x, n_chares_y);
    charm::end();
  }

  int block_width = grid_width / n_chares_x;
  int block_height = grid_height / n_chares_y;

  printf("Grid size: %d x %d\n", grid_width, grid_height);
  printf("Block size: %d x %d\n", block_width, block_height);
  printf("Chare array: %d x %d (%d total)\n", n_chares_x, n_chares_y, n_chares);
  printf("Total number of iterations: %d\n", n_iters);

  Block block;
  block_proxy->create(block, n_chares);
  constexpr int n_params = 5;
  int params[n_params] = { block_width, block_height, n_iters, n_chares_x, n_chares_y };
  for (int i = 0; i < n_chares; i++) {
    block_proxy->invoke(i, 0, params, sizeof(int) * n_params);
  }
}

__global__ void init_kernel(DataType* temperature, DataType* new_temperature,
                            int block_width, int block_height);
__global__ void pack_left_kernel(DataType* temperature, DataType* ghost,
                                 int block_width, int block_height);
__global__ void pack_right_kernel(DataType* temperature, DataType* ghost,
                                  int block_width, int block_height);
__global__ void unpack_left_kernel(DataType* temperature, DataType* ghost,
                                   int block_width, int block_height);
__global__ void unpack_right_kernel(DataType* temperature, DataType* ghost,
                                    int block_width, int block_height);
__global__ void jacobi_kernel(DataType* temperature, DataType* new_temperature,
                              int block_width, int block_height);

// Entry methods
__device__ void Block::init(void* arg) {
  int* params = (int*)arg;
  int param_idx = 0;

  // Block size and iteration count
  block_width = params[param_idx++];
  block_height = params[param_idx++];
  block_size = (unsigned long long)(block_width+2) * (unsigned long long)(block_height+2);
  n_iters = params[param_idx++];
  iter = 0;

  // Figure out this block's index and its neighbors
  int n_chares_x = params[param_idx++];
  int n_chares_y = params[param_idx++];
  int index = charm::chare::i;
  row = (index / n_chares_y);
  col = (index % n_chares_y);
  neighbor_index[LEFT] = (col == 0) ? -1 : (index-1);
  neighbor_index[RIGHT] = (col == n_chares_y-1) ? -1 : (index+1);
  neighbor_index[TOP] = (row == 0) ? -1 : (index-n_chares_y);
  neighbor_index[BOTTOM] = (row == n_chares_x-1) ? -1 : (index+n_chares_y);
  neighbor_count = 4;
  if (col == 0) neighbor_count--;
  if (col == n_chares_y-1) neighbor_count--;
  if (row == 0) neighbor_count--;
  if (row == n_chares_x-1) neighbor_count--;
  recv_count = 0;

  /*
  printf("%d: I'm (%d,%d) with %d neighbors\n", index, row, col, neighbor_count);
  printf("LEFT: %d, RIGHT: %d, TOP: %d, BOTTOM: %d\n", neighbor_index[LEFT], neighbor_index[RIGHT], neighbor_index[TOP], neighbor_index[BOTTOM]);
  */

  temperature = new DataType[block_size];
  new_temperature = new DataType[block_size];
  ghost_sizes[LEFT] = sizeof(DataType) * block_height;
  ghost_sizes[RIGHT] = sizeof(DataType) * block_height;
  ghost_sizes[TOP] = sizeof(DataType) * block_width;
  ghost_sizes[BOTTOM] = sizeof(DataType) * block_width;
  for (int i = 0; i < N_NEIGHBORS; i++) {
    boundaries[i] = new DataType[ghost_sizes[i] / sizeof(DataType) + 1]; // Need int for direction
    *(int*)boundaries[i] = (i % 2 == 0) ? (i+1) : (i-1);
  }

  dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
  dim3 grid_dim((block_height+2 + block_dim.x - 1) / block_dim.x, (block_width+2 + block_dim.y - 1) / block_dim.y);
  init_kernel<<<grid_dim, block_dim>>>(temperature, new_temperature, block_width, block_height);
  hipDeviceSynchronize();

  start_tp = cuda::std::chrono::system_clock::now();
  send_boundaries();
}

__device__ void Block::send_boundaries() {
  // Pack
  dim3 block_dim(BLOCK_DIM * BLOCK_DIM);
  dim3 grid_dim((block_height + block_dim.x - 1) / block_dim.x);
  if (neighbor_index[LEFT] != -1) {
    DataType* boundary = boundaries[LEFT]+1;
    pack_left_kernel<<<grid_dim, block_dim>>>(temperature, boundary, block_width, block_height);
  }
  if (neighbor_index[RIGHT] != -1) {
    DataType* boundary = boundaries[RIGHT]+1;
    pack_right_kernel<<<grid_dim, block_dim>>>(temperature, boundary, block_width, block_height);
  }
  hipDeviceSynchronize();

  // Send to neighbors
  if (neighbor_index[LEFT] != -1) {
    block_proxy->invoke(neighbor_index[LEFT], 1, boundaries[LEFT],
                        sizeof(DataType) + ghost_sizes[LEFT]);
  }
  if (neighbor_index[RIGHT] != -1) {
    block_proxy->invoke(neighbor_index[RIGHT], 1, boundaries[RIGHT],
                        sizeof(DataType) + ghost_sizes[RIGHT]);
  }
  if (neighbor_index[TOP] != -1) {
    memcpy(boundaries[TOP]+1,
           temperature + (block_width+2) + 1, ghost_sizes[TOP]);
    block_proxy->invoke(neighbor_index[TOP], 1, boundaries[TOP],
                        sizeof(DataType) + ghost_sizes[TOP]);
  }
  if (neighbor_index[BOTTOM] != -1) {
    memcpy(boundaries[BOTTOM]+1,
           temperature + (block_width+2) * block_height + 1, ghost_sizes[BOTTOM]);
    block_proxy->invoke(neighbor_index[BOTTOM], 1, boundaries[BOTTOM],
                        sizeof(DataType) + ghost_sizes[BOTTOM]);
  }
}

__device__ void Block::recv_ghosts(void* arg) {
  int dir = *(int*)arg;
  DataType* ghost = (DataType*)arg + 1;
  dim3 block_dim(BLOCK_DIM * BLOCK_DIM);
  dim3 grid_dim((block_height + (block_dim.x-1)) / block_dim.x);

  switch (dir) {
    case LEFT:
      unpack_left_kernel<<<grid_dim, block_dim>>>(temperature, ghost, block_width, block_height);
      hipDeviceSynchronize();
      break;
    case RIGHT:
      unpack_right_kernel<<<grid_dim, block_dim>>>(temperature, ghost, block_width, block_height);
      hipDeviceSynchronize();
      break;
    case TOP:
      memcpy(temperature + 1, ghost, ghost_sizes[TOP]);
      break;
    case BOTTOM:
      memcpy(temperature + (block_width+2) * (block_height+1) + 1, ghost, ghost_sizes[BOTTOM]);
      break;
    default:
      printf("Wrong direction!\n");
      break;
  }

  if (++recv_count == neighbor_count) {
    recv_count = 0;
    update();
  }
}

__device__ void Block::update() {
  dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
  dim3 grid_dim((block_height + (block_dim.x-1)) / block_dim.x, (block_width + (block_dim.y-1)) / block_dim.y);
  jacobi_kernel<<<grid_dim, block_dim>>>(temperature, new_temperature, block_width, block_height);
  hipDeviceSynchronize();

  /*
  printf("OLD\n");
  for (int i = 0; i < block_height+2; i++) {
    for (int j = 0; j < block_width+2; j++) {
      printf("%-10.3lf ", temperature[IDX(i,j)]);
    }
    printf("\n");
  }

  printf("NEW\n");
  for (int i = 0; i < block_height+2; i++) {
    for (int j = 0; j < block_width+2; j++) {
      printf("%-10.3lf ", new_temperature[IDX(i,j)]);
    }
    printf("\n");
  }
  */

  if (++iter == n_iters) {
    end_tp = cuda::std::chrono::system_clock::now();
    cuda::std::chrono::duration<double> diff = end_tp - start_tp;
    printf("Chare (%d,%d) completed %d iterations in %.6lf seconds\n", row, col, iter, diff.count());

    charm::end();
  } else {
    DataType* tmp = temperature;
    temperature = new_temperature;
    new_temperature = tmp;

    send_boundaries();
  }
}

// GPU kernels
__global__ void init_kernel(DataType* temperature, DataType* new_temperature,
                            int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i > 0 && i < block_height+1 && j > 0 && j < block_width+1) {
    temperature[IDX(i,j)] = 0;
    new_temperature[IDX(i,j)] = 0;
  } else if (i < block_height+2 && j < block_width+2) {
    temperature[IDX(i,j)] = 10;
    new_temperature[IDX(i,j)] = 10;
  }
}

__global__ void pack_left_kernel(DataType* temperature, DataType* ghost,
                                 int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < block_height+1) {
    ghost[i-1] = temperature[IDX(i,1)];
  }
}

__global__ void pack_right_kernel(DataType* temperature, DataType* ghost,
                                  int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < block_height+1) {
    ghost[i-1] = temperature[IDX(i,block_width)];
  }
}

__global__ void unpack_left_kernel(DataType* temperature, DataType* ghost,
                                   int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < block_height+1) {
    temperature[IDX(i,0)] = ghost[i-1];
  }
}

__global__ void unpack_right_kernel(DataType* temperature, DataType* ghost,
                                    int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < block_height+1) {
    temperature[IDX(i,block_width+1)] = ghost[i-1];
  }
}

__global__ void jacobi_kernel(DataType* temperature, DataType* new_temperature,
                              int block_width, int block_height) {
  int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int j = blockDim.y * blockIdx.y + threadIdx.y + 1;
  if (i < block_height+1 && j < block_width+1) {
    new_temperature[IDX(i,j)] = (temperature[IDX(i,j)] + temperature[IDX(i,j-1)]
      + temperature[IDX(i,j+1)] + temperature[IDX(i-1,j)] + temperature[IDX(i+1,j)]) * 0.2;
  }
}
