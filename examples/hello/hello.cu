#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hello.h"

__shared__ charm::chare_proxy<Hello>* hello_proxy;

__device__ void charm::create_chares(int argc, char** argv, size_t* argvs) {
  // Register Hello chare and its entry methods
  hello_proxy = new charm::chare_proxy<Hello>();
  hello_proxy->add_entry_method<&entry_greet>();
  hello_proxy->create(charm::n_pes() * 2);
}

__device__ void charm::main(int argc, char** argv, size_t* argvs) {
  __shared__ int send_int;

  if (threadIdx.x == 0) {
    send_int = 0;
  }
  __syncthreads();

  // Send integer to first chare
  hello_proxy->invoke(0, 0, &send_int, sizeof(int));
}

__device__ void Hello::greet(void* arg) {
  int recv_int = ((int*)arg)[0];
  __shared__ int send_int;
  int i = charm::chare::i;
  int n = charm::chare::n;
  if (threadIdx.x == 0) {
    printf("Hello I'm %d of %d! Received %d\n", i, n, recv_int);
  }

  if (i == n-1) {
    charm::end();
  } else {
    if (threadIdx.x == 0) {
      send_int = recv_int + 1;
    }
    __syncthreads();

    hello_proxy->invoke(i + 1, 0, &send_int, sizeof(int));
  }
}
