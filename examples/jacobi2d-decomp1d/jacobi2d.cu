#include "hip/hip_runtime.h"
#include <stdio.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include "jacobi2d.h"

#define PI 3.141592

#define GRID_WIDTH 16384
#define GRID_HEIGHT 16384
#define N_ITERS 1000

#ifdef SM_LEVEL
// SM-level scheduling
#define GID (threadIdx.x)
#define GROUP_SIZE (blockDim.x)
#define BARRIER_LOCAL __syncthreads()
__shared__ charm::chare_proxy<Block>* block_proxy;
#else
// GPU-level scheduling
#define GID (blockDim.x * blockIdx.x + threadIdx.x)
#define GROUP_SIZE (gridDim.x * blockDim.x)
#define BARRIER_LOCAL charm::barrier_local()
__device__ charm::chare_proxy<Block>* block_proxy;
__device__ int* params;
__managed__ real* a_global;
__managed__ real* a_new_global;
__managed__ uint64_t* sync_arr_global;
#endif

void charm::main_host(int argc, char** argv) {
  assert(argc == 5);
  int n_chares = atoi(argv[1]);
  int nx = atoi(argv[2]);
  int ny = atoi(argv[3]);
  int iter_max = atoi(argv[4]);
  int pe = nvshmem_my_pe();
  if (pe == 0) {
    printf("Jacobi2D: Chares %d, nx %d, ny %d, iter_max %d\n",
        n_chares, nx, ny, iter_max);
  }

  // Compute chunk size and allocate memory
  int n_pes = nvshmem_n_pes();
  assert(n_chares % n_pes == 0);
  int n_chares_per_pe = n_chares / n_pes;
  int chunk_size_low = (ny - 2) / n_chares;
  int chunk_size_high = chunk_size_low + 1;
  size_t a_global_size = nx * (chunk_size_high + 2) * sizeof(real) * n_chares_per_pe;
  size_t a_new_global_size = a_global_size;
  size_t sync_global_size = 2 * sizeof(uint64_t) * n_chares_per_pe;

  a_global = (real*)nvshmem_malloc(a_global_size);
  a_new_global = (real*)nvshmem_malloc(a_new_global_size);
  sync_arr_global = (uint64_t*)nvshmem_malloc(sync_global_size);
  assert(a_global && a_new_global && sync_arr_global);
}

__device__ void charm::main(int argc, char** argv, size_t* argvs, int pe) {
  // Execute on all elements

  // Process command line arguments
  int n_chares = charm::n_pes();
  int grid_width = GRID_WIDTH;
  int grid_height = GRID_HEIGHT;
  int n_iters = N_ITERS;
  if (argc >= 2) n_chares = charm::device_atoi(argv[1], argvs[1]);
  if (argc >= 3) grid_width = charm::device_atoi(argv[2], argvs[2]);
  if (argc >= 4) grid_height = charm::device_atoi(argv[3], argvs[3]);
  if (argc >= 5) n_iters = charm::device_atoi(argv[4], argvs[4]);

  if (GID == 0) {
    // Create chare proxy and register entry methods
    block_proxy = new charm::chare_proxy<Block>();
    block_proxy->add_entry_method<&entry_init>();
    /*
    block_proxy->add_entry_method<&entry_recv_halo>();
    block_proxy->add_entry_method<&entry_terminate>();
    */

    // Create chares
    block_proxy->create(n_chares);
  }
  BARRIER_LOCAL;

  barrier();

  // Executed only on PE 0
  if (pe == 0) {
    constexpr int n_params = 3;
#ifdef SM_LEVEL
    __shared__ int params[n_params];
#endif

    if (GID == 0) {
#ifndef SM_LEVEL
      params = new int[n_params];
#endif

      params[0] = grid_width;
      params[1] = grid_height;
      params[2] = n_iters;

      printf("Chares: %d\n", n_chares);
      printf("Grid size: %d x %d\n", grid_width, grid_height);
      printf("Iterations: %d\n", n_iters);
    }
    BARRIER_LOCAL;

    block_proxy->invoke_all(0, params, sizeof(int) * n_params);
  }
}

__device__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, int ny);

__device__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              const int iy_start, const int iy_end, const int nx, const int top_pe,
                              const int top_iy, const int bottom_pe, const int bottom_iy,
                              const size_t a_count, const int npes_per_gpu);

__device__ void syncneighborhood_kernel(int my_pe, int num_pes, uint64_t* sync_arr,
                                        long counter, size_t sync_count, int npes_per_gpu);

// Entry methods
__device__ void Block::init(void* arg) {
  if (GID == 0) {
    int* params = (int*)arg;
    nx = params[0];
    ny = params[1];
    iter_max = params[2];
    npes = charm::chare::n;
    mype = charm::chare::i;
    recv_count = 0;
    term_count = 0;
    printf("Block %3d init on PE %d\n", mype, charm::my_pe());

    // Compute chunk size and allocate memory
    int chunk_size_low = (ny - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;
    int num_ranks_low = npes * chunk_size_low + npes - (ny - 2);
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    ngpus = charm::n_pes();
    npes_per_gpu = npes / ngpus;
    mype_local = mype % npes_per_gpu;
    a_count = nx * (chunk_size_high + 2);
    a_size = a_new_size = a_count * sizeof(real);
    a = a_global + a_count * mype_local;
    a_new = a_new_global + a_count * mype_local;
    assert(a && a_new);

    // Signal array for neighborhood synchronization
    sync_count = 2;
    sync_size = sync_count * sizeof(uint64_t);
    sync_arr = sync_arr_global + sync_count * mype_local;
    synccounter = 1;
    assert(sync_arr);

    // Calculate local domain boundaries
    if (mype < num_ranks_low) {
        iy_start_global = mype * chunk_size_low + 1;
    } else {
        iy_start_global =
            num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high + 1;
    }
    iy_end_global = iy_start_global + chunk_size - 1;
    iy_end_global = min(iy_end_global, ny - 4); // Do not process boundaries

    iy_start = 1;
    iy_end = (iy_end_global - iy_start_global + 1) + iy_start;

    // Calculate boundary indices for top and bottom boundaries
    top_pe = mype > 0 ? mype - 1 : (npes - 1);
    bottom_pe = (mype + 1) % npes;

    iy_end_top = (top_pe < num_ranks_low) ? chunk_size_low + 1 : chunk_size_high + 1;
    iy_start_bottom = 0;

    // Set initial reference number
    //block_proxy->set_refnum(mype, iter);
  }
  BARRIER_LOCAL;

#ifdef SM_LEVEL
  charm::memset_kernel_block(a, 0, a_size);
  charm::memset_kernel_block(a_new, 0, a_new_size);
  charm::memset_kernel_block(sync_arr, 0, sync_size);
#else
  charm::memset_kernel_grid(a, 0, a_size);
  charm::memset_kernel_grid(a_new, 0, a_new_size);
  charm::memset_kernel_grid(sync_arr, 0, sync_size);
#endif

  initialize_boundaries(a_new, a, PI, iy_start_global - 1, nx, chunk_size, ny - 2);

  // Start iteration
  iterate();
}

__device__ void Block::iterate() {
  if (GID == 0) {
    start_tp = cuda::std::chrono::system_clock::now();
  }
  BARRIER_LOCAL;

  for (int iter = 0; iter < iter_max; iter++) {
    // Execute Jacobi update kernel
    jacobi_kernel(a_new, a, iy_start, iy_end, nx, top_pe, iy_end_top,
        bottom_pe, iy_start_bottom, a_count, npes_per_gpu);
    BARRIER_LOCAL;

    // Neighborhood synchronization
    if (GID == 0) {
      syncneighborhood_kernel(mype, npes, sync_arr, synccounter, sync_count, npes_per_gpu);
      synccounter++;
    }
    BARRIER_LOCAL;

    // Swap pointers
    /*
    if (GID == 0) {
      real* temp = a;
      a = a_new;
      a_new = temp;
    }
    BARRIER_LOCAL;
    */
  }

  if (GID == 0) {
    end_tp = cuda::std::chrono::system_clock::now();
    cuda_dur diff = end_tp - start_tp;
    printf("Block %3d completed %4d iterations in %lf s\n", mype, iter_max, diff.count());
  }
  BARRIER_LOCAL;
}

/*
__device__ void Block::update() {
  // Execute Jacobi update kernel
  jacobi_kernel(a_new, a, iy_start, iy_end, nx);

  // Send halo to neighbors
  send_halo();
}

__device__ void Block::send_halo() {
  block_proxy->invoke(top_pe, 1, a_new + iy_start * nx, nx * sizeof(real), iter);
  block_proxy->invoke(bottom_pe, 1, a_new + (iy_end - 1) * nx, nx * sizeof(real), iter);
}

__device__ void Block::recv_halo(void* arg) {
  // TODO: Figure out if halo came from the top or bottom neighbor & memcpy
#ifdef SM_LEVEL
  __shared__ bool done;
  __shared__ bool end;
#endif

  if (GID == 0) {
    done = false;
    end = false;

    if (++recv_count == 2) {
      // Received halos from both neighbors
      printf("Chare %3d completed iteration %d\n", mype, iter);
      recv_count = 0;
      done = true;

      // Set reference number for next iteration
      iter++;
      block_proxy->set_refnum(mype, iter);

      if (iter == iter_max) {
        end = true;
        printf("Chare %3d completed all %d iterations\n", mype, iter_max);
      }
    }
  }
  BARRIER_LOCAL;

  if (done) {
    if (end) {
      block_proxy->invoke(0, 2, nullptr, 0, -1);
    } else {
      update();
    }
  }
}

__device__ void Block::terminate(void* arg) {
  // Terminate only when all chares have finished
  if (GID == 0) {
    term_count++;
  }
  BARRIER_LOCAL;

  if (term_count == npes) {
    charm::end();
  }
}
*/

__device__ void initialize_boundaries(real* __restrict__ const a_new, real* __restrict__ const a,
                                      const real pi, const int offset, const int nx,
                                      const int my_ny, int ny) {
  for (int iy = GID; iy < my_ny; iy += GROUP_SIZE) {
    const real y0 = sin(2.0 * pi * (offset + iy) / (ny - 1));
    a[(iy + 1) * nx + 0] = y0;
    a[(iy + 1) * nx + (nx - 1)] = y0;
    a_new[(iy + 1) * nx + 0] = y0;
    a_new[(iy + 1) * nx + (nx - 1)] = y0;
  }
}

__device__ void jacobi_kernel(real* __restrict__ const a_new, const real* __restrict__ const a,
                              const int iy_start, const int iy_end, const int nx, const int top_pe,
                              const int top_iy, const int bottom_pe, const int bottom_iy,
                              const size_t a_count, const int npes_per_gpu) {
  for (int iy = iy_start; iy < iy_end; iy++) {
    for (int ix = GID + 1; ix < (nx - 1); ix += GROUP_SIZE) {
      const real new_val = 0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] +
                                   a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
      a_new[iy * nx + ix] = new_val;
    }
  }

  BARRIER_LOCAL;

  /* Communicate the boundaries */
  int top_gpu = top_pe / npes_per_gpu;
  int bottom_gpu = bottom_pe / npes_per_gpu;
  int top_pe_local = top_pe % npes_per_gpu;
  int bottom_pe_local = bottom_pe % npes_per_gpu;
  real* a_new_top = a_new_global + a_count * top_pe_local;
  real* a_new_bottom = a_new_global + a_count * bottom_pe_local;
  for (int block_ix = blockIdx.x * blockDim.x + 1; block_ix < ((nx + blockDim.x - 1) / blockDim.x);
      block_ix += gridDim.x) {
    nvshmemx_float_put_nbi_block(a_new_top + top_iy * nx + block_ix, a_new + iy_start * nx + block_ix,
                                 min(blockDim.x, nx - 1 - block_ix), top_gpu);
    nvshmemx_float_put_nbi_block(a_new_bottom + bottom_iy * nx + block_ix,
                                 a_new + (iy_end - 1) * nx + block_ix,
                                 min(blockDim.x, nx - 1 - block_ix), bottom_gpu);
  }
}

__device__ void syncneighborhood_kernel(int my_pe, int num_pes, uint64_t* sync_arr,
                                        long counter, size_t sync_count, int npes_per_gpu) {
    int next_rank = (my_pe + 1) % num_pes;
    int prev_rank = (my_pe == 0) ? num_pes - 1 : my_pe - 1;
    nvshmem_quiet(); /* To ensure all prior nvshmem operations have been completed */

    /* Notify neighbors about arrival */
    int next_gpu = next_rank / npes_per_gpu;
    int prev_gpu = prev_rank / npes_per_gpu;
    int next_rank_local = next_rank % npes_per_gpu;
    int prev_rank_local = prev_rank % npes_per_gpu;
    uint64_t* sync_arr_next = sync_arr_global + sync_count * next_rank_local;
    uint64_t* sync_arr_prev = sync_arr_global + sync_count * prev_rank_local;
    /*
    printf("Block %d signaling %d (GPU %d, %p) and %d (GPU %d, %p)\n", my_pe,
        next_rank, next_gpu, sync_arr_next, prev_rank, prev_gpu, sync_arr_prev + 1);
        */
    nvshmemx_signal_op(sync_arr_next, counter, NVSHMEM_SIGNAL_SET, next_gpu);
    nvshmemx_signal_op(sync_arr_prev + 1, counter, NVSHMEM_SIGNAL_SET, prev_gpu);

    /* Wait for neighbors notification */
    //printf("Block %d waiting on signals at %p and %p\n", my_pe, sync_arr, sync_arr + 1);
    nvshmem_uint64_wait_until_all(sync_arr, 2, NULL, NVSHMEM_CMP_GE, counter);
}
