#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include "Message.h"
#include "user.h"
#include "nvcharm.h"

#define DEBUG 0

/*
// FIXME: Hard-coded limits
#define EM_CNT_MAX 1024 // Maximum number of entry methods
#define SM_CNT 80 // Number of SMs
#define MSG_CNT_MAX 1024 // Maximum number of messages in message queue
#define MSG_IDX(sm,idx) (MSG_CNT_MAX*(sm) + (idx))
#define CHARE_CNT_MAX 1024 // Maxinum number of chare types

__device__ ChareType* chare_types[SM_CNT * CHARE_CNT_MAX];
__device__ int chare_cnt[SM_CNT];
__device__ EntryMethod* entry_methods[EM_CNT_MAX];
__device__ Message* msg_queue[SM_CNT * MSG_CNT_MAX];
__device__ int msg_cnt[SM_CNT];
__device__ int terminate[SM_CNT];

__device__ uint get_smid() {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

using clock_value_t = long long;

__device__ void sleep(clock_value_t sleep_cycles) {
  clock_value_t start = clock64();
  clock_value_t cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}

__device__ void send(int sm, Message* msg) {
  int msg_idx = atomicAdd(&msg_cnt[sm], 1);
  msg_queue[MSG_IDX(sm,msg_idx)] = msg;
#if DEBUG
  printf("Stored message in idx %d, msg %p\n", MSG_IDX(sm,msg_idx), msg);
#endif
}

__device__ void recv(int my_sm, int& processed, bool& terminate) {
  // TODO: Recv doesn't happen without follownig print statement, why?
#if DEBUG
  printf("SM %d checking idx %d\n", my_sm, MSG_IDX(my_sm, processed));
#endif
  Message*& msg = msg_queue[MSG_IDX(my_sm, processed)];
  if (msg) {
#if DEBUG
    printf("SM %d received message %p, SM %d, ep %d\n",
        my_sm, msg, msg->src_sm, msg->ep);
#endif

    if (msg->ep == -1) {
      terminate = true;
#if DEBUG
      printf("SM %d terminating\n", my_sm);
#endif
    }

    // Handle received message
    entry_methods[msg->ep]->call();

    msg = nullptr;
    processed++;
  }
}

__global__ void scheduler(DeviceCtx* ctx) {
  //const int my_sm = get_smid();
  const int my_sm = blockIdx.x;
  __shared__ int processed;
  __shared__ bool terminate;

  register_entry_methods(entry_methods);

  // Leader thread in each thread block runs the scheduler loop
  if (threadIdx.x == 0) {
    processed = 0;
    terminate = false;

    if (blockIdx.x == 0) {
      printf("SMs: %d\n", ctx->n_sms);

      // Execute user's main function
      charm_main();
    }

    // Scheduler loop
    do {
      recv(my_sm, processed, terminate);
      //sleep(1);
    } while (!terminate);
  }
}
*/

__global__ void simple_shift(int *destination) {
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int peer = (mype + 1) % npes;
  printf("nvshmem pes: %d\n", npes);

  nvshmem_int_p(destination, mype, peer);
  //nvshmem_barrier_all();
}

int main(int argc, char* argv[]) {
  int rank, msg;
  hipStream_t stream;

  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  /*
  // Print GPU device properties
  int device = 0;
  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  printf("* GPU properties\n"
      "Name: %s\nCompute capability: %d.%d\nSMs: %d\n"
      "Max threads per SM: %d\nKernel runtime limit: %d\n"
      "Managed memory support: %d\nCooperative kernel support: %d\n\n",
      prop.name, prop.major, prop.minor, prop.multiProcessorCount,
      prop.maxThreadsPerMultiProcessor, prop.kernelExecTimeoutEnabled,
      prop.managedMemory, prop.cooperativeLaunch);

  if (!prop.managedMemory) {
    fprintf(stderr, "Managed memory support required\n");
    exit(1);
  }
  */

  hipStreamCreate(&stream);

  int *destination = (int *) nvshmem_malloc(sizeof(int));

  void* args[1] = {&destination};
  nvshmemx_collective_launch((const void*)simple_shift, 1, 1, args, 0, stream);
  //simple_shift<<<1, 1, 0, stream>>>(destination);
  //nvshmemx_barrier_all_on_stream(stream);
  nvshmem_barrier_all();
  hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

  hipStreamSynchronize(stream);
  printf("%d: received message %d\n", nvshmem_my_pe(), msg);

  nvshmem_free(destination);
  nvshmem_finalize();
  MPI_Finalize();

  /*
  // Create device context
  DeviceCtx* ctx;
  hipMallocManaged(&ctx, sizeof(DeviceCtx));
  ctx->n_sms = prop.multiProcessorCount;

  // Obtain kernel block and grid sizes
  int block_size = 1;
  int grid_size = ctx->n_sms;
  if (argc > 1) block_size = atoi(argv[1]);
  if (argc > 2) grid_size = atoi(argv[2]);
  printf("* Test properties\n"
      "Block size: %d\nGrid size: %d\n\n", block_size, grid_size);

  // Run kernel
  scheduler<<<grid_size, block_size>>>(ctx);
  hipDeviceSynchronize();
  */

  return 0;
}

/******************** Chare ********************/

/*
template <typename T>
__device__ Chare<T>::Chare(T obj_, int n_chares_) : obj(obj_), n_chares(n_chares_) {
  // Create chare objects on all GPUs
  // TODO: Assume 1 GPU for now
  mapping = new Mapping[SM_CNT];
  int rem = n_chares % SM_CNT;
  int start_idx = 0;
  for (int i = 0; i < SM_CNT; i++) {
    int n_chares_sm = n_chares / SM_CNT;
    if (i < rem) n_chares_sm++;
    mapping[i].sm_id = i;
    mapping[i].start_idx = start_idx;
    mapping[i].end_idx = start_idx + n_chares_sm - 1;
    start_idx += n_chares_sm;

    CreationMessage<T>* create_msg = new CreationMessage<T>(obj);
  }
}

template <typename T>
__device__ void Chare<T>::invoke(int ep, int idx) {
  if (idx == -1) {
    // Broadcast to all chares
    for (int i = 0; i < n_chares; i++) {
      Message* msg = (Message*)malloc(sizeof(Message));
      msg->ep = ep;
      int target_sm = mapping[i];
      send(target_sm, msg);
    }
  } else {
    // P2P
    Message* msg = (Message*)malloc(sizeof(Message));
    msg->ep = ep;
    int target_sm = mapping[idx];
    send(target_sm, msg);
  }
}
*/
