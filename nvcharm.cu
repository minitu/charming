#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include "Message.h"
#include "user.h"
#include "nvcharm.h"

#define DEBUG 0

#define MSG_CNT_MAX 1e6 // Maximum number of messages per PE
#define EM_CNT_MAX 1024 // Maximum number of entry methods

__device__ Message* msg_queue_symbol;
__device__ int* msg_queue_head_symbol;
__device__ int* msg_queue_tail_symbol;
__device__ EntryMethod* entry_methods[EM_CNT_MAX];

__device__ inline void send(int dst_pe, Message* msg) {
  /*
  int msg_idx = atomicAdd(&msg_cnt[sm], 1);
  msg_queue[MSG_IDX(sm,msg_idx)] = msg;
  */
}

__device__ inline void recv(int my_pe, bool& terminate) {
  /*
  int* head_ptr = nvshmem_ptr(msg_queue_tail_symbol, my_pe);
  if (msg) {

    if (msg->ep == -1) {
      terminate = true;
    }

    // Handle received message
    entry_methods[msg->ep]->call();

    msg = nullptr;
    processed++;
  }
  */
}

/*
// FIXME: Hard-coded limits
#define EM_CNT_MAX 1024 // Maximum number of entry methods
#define SM_CNT 80 // Number of SMs
#define MSG_CNT_MAX 1024 // Maximum number of messages in message queue
#define MSG_IDX(sm,idx) (MSG_CNT_MAX*(sm) + (idx))
#define CHARE_CNT_MAX 1024 // Maxinum number of chare types

__device__ ChareType* chare_types[SM_CNT * CHARE_CNT_MAX];
__device__ int chare_cnt[SM_CNT];
__device__ EntryMethod* entry_methods[EM_CNT_MAX];
__device__ Message* msg_queue[SM_CNT * MSG_CNT_MAX];
__device__ int msg_cnt[SM_CNT];
__device__ int terminate[SM_CNT];

__device__ uint get_smid() {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

using clock_value_t = long long;

__device__ void sleep(clock_value_t sleep_cycles) {
  clock_value_t start = clock64();
  clock_value_t cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}

__device__ void send(int sm, Message* msg) {
  int msg_idx = atomicAdd(&msg_cnt[sm], 1);
  msg_queue[MSG_IDX(sm,msg_idx)] = msg;
#if DEBUG
  printf("Stored message in idx %d, msg %p\n", MSG_IDX(sm,msg_idx), msg);
#endif
}

__device__ void recv(int my_sm, int& processed, bool& terminate) {
  // TODO: Recv doesn't happen without follownig print statement, why?
#if DEBUG
  printf("SM %d checking idx %d\n", my_sm, MSG_IDX(my_sm, processed));
#endif
  Message*& msg = msg_queue[MSG_IDX(my_sm, processed)];
  if (msg) {
#if DEBUG
    printf("SM %d received message %p, SM %d, ep %d\n",
        my_sm, msg, msg->src_sm, msg->ep);
#endif

    if (msg->ep == -1) {
      terminate = true;
#if DEBUG
      printf("SM %d terminating\n", my_sm);
#endif
    }

    // Handle received message
    entry_methods[msg->ep]->call();

    msg = nullptr;
    processed++;
  }
}

__global__ void scheduler(DeviceCtx* ctx) {
  //const int my_sm = get_smid();
  const int my_sm = blockIdx.x;
  __shared__ int processed;
  __shared__ bool terminate;

  register_entry_methods(entry_methods);

  // Leader thread in each thread block runs the scheduler loop
  if (threadIdx.x == 0) {
    processed = 0;
    terminate = false;

    if (blockIdx.x == 0) {
      printf("SMs: %d\n", ctx->n_sms);

      // Execute user's main function
      charm_main();
    }

    // Scheduler loop
    do {
      recv(my_sm, processed, terminate);
      //sleep(1);
    } while (!terminate);
  }
}
*/

/*
__global__ void simple_shift(int *destination) {
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int peer = (mype + 1) % npes;

  if (!blockIdx.x && !threadIdx.x) {
    nvshmem_int_p(destination, mype, peer);
    nvshmem_barrier_all();
  }
}
*/

__global__ void scheduler(Message* msg_queue, int* msg_queue_head, int* msg_queue_tail) {
  if (!blockIdx.x && !threadIdx.x) {
    int my_pe = nvshmem_my_pe();
    int n_pes = nvshmem_n_pes();
    bool terminate = false;

    // Register user's entry methods
    register_entry_methods(entry_methods);

    // Execute user's main function
    charm_main();

    // Scheduler loop
    do {
      recv(my_pe, terminate);
    } while (!terminate);
  }
}

int main(int argc, char* argv[]) {
  int rank, msg;
  hipStream_t stream;

  // Initialize MPI
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Initialize NVSHMEM
  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  // Initialize CUDA
  hipSetDevice(0);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // NVSHMEM symmetric memory allocations
  Message* msg_queue = (Message*) nvshmem_malloc(sizeof(Message) * MSG_CNT_MAX);
  int* msg_queue_head = (int*) nvshmem_malloc(sizeof(int));
  int* msg_queue_tail = (int*) nvshmem_malloc(sizeof(int));
  nvshmem_barrier_all();
  hipMemcpyToSymbol(HIP_SYMBOL(msg_queue_symbol), msg_queue, sizeof(Message*));
  hipMemcpyToSymbol(HIP_SYMBOL(msg_queue_head_symbol), msg_queue_head, sizeof(int*));
  hipMemcpyToSymbol(HIP_SYMBOL(msg_queue_tail_symbol), msg_queue_tail, sizeof(int*));

  // Launch scheduler
  int grid_size = (argc > 1) ? atoi(argv[1]) : 1;
  int block_size = (argc > 2) ? atoi(argv[2]) : 1;
  if (!rank) {
    printf("NVCHARM\nGrid size: %d\nBlock size: %d\n", grid_size, block_size);
  }
  void* scheduler_args[3] = { &msg_queue, &msg_queue_head, &msg_queue_tail };
  nvshmemx_collective_launch((const void*)scheduler, grid_size, block_size,
      scheduler_args, 0, stream);
  cuda_check_error();
  hipStreamSynchronize(stream);
  //nvshmemx_barrier_all_on_stream(stream); // Hangs
  nvshmem_barrier_all();

  // Finalize NVSHMEM and MPI
  nvshmem_free(msg_queue);
  nvshmem_finalize();
  hipStreamDestroy(stream);
  MPI_Finalize();

  return 0;
}

template <typename T>
__device__ Chare<T>::Chare(T obj_, int n_chares_) : obj(obj_), n_chares(n_chares_) {
  /*
  // TODO: Create chare objects on all GPUs
  mapping = new Mapping[SM_CNT];
  int rem = n_chares % SM_CNT;
  int start_idx = 0;
  for (int i = 0; i < SM_CNT; i++) {
    int n_chares_sm = n_chares / SM_CNT;
    if (i < rem) n_chares_sm++;
    mapping[i].sm_id = i;
    mapping[i].start_idx = start_idx;
    mapping[i].end_idx = start_idx + n_chares_sm - 1;
    start_idx += n_chares_sm;

    //CreationMessage<T>* create_msg = new CreationMessage<T>(obj);
  }
  */
}

template <typename T>
__device__ void Chare<T>::invoke(int ep, int idx) {
  /*
  if (idx == -1) {
    // Broadcast to all chares
    for (int i = 0; i < n_chares; i++) {
      Message* msg = (Message*)malloc(sizeof(Message));
      msg->ep = ep;
      int target_sm = mapping[i];
      send(target_sm, msg);
    }
  } else {
    // P2P
    Message* msg = (Message*)malloc(sizeof(Message));
    msg->ep = ep;
    int target_sm = mapping[idx];
    send(target_sm, msg);
  }
  */
}
