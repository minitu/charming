#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <mpi.h>
#include "Message.h"
#include "user.h"
#include "nvcharm.h"
#include "ringbuf.h"

#define DEBUG 0

#define MSG_CNT_MAX 1e6 // Maximum number of messages per PE
#define EM_CNT_MAX 1024 // Maximum number of entry methods

__device__ EntryMethod* entry_methods[EM_CNT_MAX];

/*
__device__ inline void send(int dst_pe, Message* msg) {
  int msg_idx = atomicAdd(&msg_cnt[sm], 1);
  msg_queue[MSG_IDX(sm,msg_idx)] = msg;
}

__device__ inline void recv(int my_pe, bool& terminate) {
  /*
  int* head_ptr = nvshmem_ptr(msg_queue_tail_symbol, my_pe);
  if (msg) {

    if (msg->ep == -1) {
      terminate = true;
    }

    // Handle received message
    entry_methods[msg->ep]->call();

    msg = nullptr;
    processed++;
  }
}

// FIXME: Hard-coded limits
#define EM_CNT_MAX 1024 // Maximum number of entry methods
#define SM_CNT 80 // Number of SMs
#define MSG_CNT_MAX 1024 // Maximum number of messages in message queue
#define MSG_IDX(sm,idx) (MSG_CNT_MAX*(sm) + (idx))
#define CHARE_CNT_MAX 1024 // Maxinum number of chare types

__device__ ChareType* chare_types[SM_CNT * CHARE_CNT_MAX];
__device__ int chare_cnt[SM_CNT];
__device__ EntryMethod* entry_methods[EM_CNT_MAX];
__device__ Message* msg_queue[SM_CNT * MSG_CNT_MAX];
__device__ int msg_cnt[SM_CNT];
__device__ int terminate[SM_CNT];

__device__ uint get_smid() {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

using clock_value_t = long long;

__device__ void sleep(clock_value_t sleep_cycles) {
  clock_value_t start = clock64();
  clock_value_t cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}

__device__ void send(int sm, Message* msg) {
  int msg_idx = atomicAdd(&msg_cnt[sm], 1);
  msg_queue[MSG_IDX(sm,msg_idx)] = msg;
#if DEBUG
  printf("Stored message in idx %d, msg %p\n", MSG_IDX(sm,msg_idx), msg);
#endif
}

__device__ void recv(int my_sm, int& processed, bool& terminate) {
  // TODO: Recv doesn't happen without follownig print statement, why?
#if DEBUG
  printf("SM %d checking idx %d\n", my_sm, MSG_IDX(my_sm, processed));
#endif
  Message*& msg = msg_queue[MSG_IDX(my_sm, processed)];
  if (msg) {
#if DEBUG
    printf("SM %d received message %p, SM %d, ep %d\n",
        my_sm, msg, msg->src_sm, msg->ep);
#endif

    if (msg->ep == -1) {
      terminate = true;
#if DEBUG
      printf("SM %d terminating\n", my_sm);
#endif
    }

    // Handle received message
    entry_methods[msg->ep]->call();

    msg = nullptr;
    processed++;
  }
}

__global__ void scheduler(DeviceCtx* ctx) {
  //const int my_sm = get_smid();
  const int my_sm = blockIdx.x;
  __shared__ int processed;
  __shared__ bool terminate;

  register_entry_methods(entry_methods);

  // Leader thread in each thread block runs the scheduler loop
  if (threadIdx.x == 0) {
    processed = 0;
    terminate = false;

    if (blockIdx.x == 0) {
      printf("SMs: %d\n", ctx->n_sms);

      // Execute user's main function
      charm_main();
    }

    // Scheduler loop
    do {
      recv(my_sm, processed, terminate);
      //sleep(1);
    } while (!terminate);
  }
}

__global__ void simple_shift(int *destination) {
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int peer = (mype + 1) % npes;

  if (!blockIdx.x && !threadIdx.x) {
    nvshmem_int_p(destination, mype, peer);
    nvshmem_barrier_all();
  }
}
*/

struct Message {
  int i;
  char c;

  __device__ Message(int i_, char c_) : i(i_), c(c_) {}
};

__global__ void scheduler(ringbuf_t* rbuf, size_t rbuf_size,
                          single_ringbuf_t* mbuf, size_t mbuf_size) {
  if (!blockIdx.x && !threadIdx.x) {
    int my_pe = nvshmem_my_pe();
    int n_pes = nvshmem_n_pes();
    bool terminate = false;

    // Register user's entry methods
    register_entry_methods(entry_methods);

    // Execute user's main function
    charm_main();

    // Initialize message queue
    ringbuf_init(rbuf, rbuf_size);
    single_ringbuf_init(mbuf, mbuf_size);

    nvshmem_barrier_all();

    ringbuf_off_t rret, mret;
    int dst_pe = 0;
    if (my_pe) {
      /*
      // Secure region in destination PE's message queue
      rret = ringbuf_acquire(rbuf, sizeof(Message), dst_pe);
      assert(rret != -1 && rret < rbuf_size);
      printf("PE %d: acquired %llu\n", my_pe, rret);

      // Secure region in my message pool
      mret = single_ringbuf_acquire(mbuf, sizeof(Message));
      assert(mret != -1 && mret < mbuf_size);

      // Populate message
      Message* msg = (Message*)msg_buf;
      msg->i = my_pe;
      msg->c = 'a';

      // Send message
      nvshmem_char_put((char*)rbuf->ptr + ret, (char*)msg, sizeof(Message), dst_pe);
      single_ringbuf_produce(mbuf);
      nvshmem_quiet();
      ringbuf_produce(rbuf, dst_pe);
      */
    } else {
      // TODO
      for (int i = 0; i < 3; i++) {
        mret = single_ringbuf_acquire(mbuf, 32);
        printf("acquired %lld\n", mret);
        single_ringbuf_produce(mbuf);
      }
      for (int i = 0; i < 2; i++) {
        size_t offset;
        mret = single_ringbuf_consume(mbuf, &offset);
        printf("consumed %lld, size %lld, release %d\n", offset, mret, 32);
        single_ringbuf_release(mbuf, 32);
      }
      for (int i = 0; i < 32; i++) {
        mret = single_ringbuf_acquire(mbuf, 4);
        printf("acquired %lld\n", mret);
      }
    }

    // Scheduler loop
    /*
    do {
      recv(my_pe, terminate);
    } while (!terminate);
    */
  }
}

int main(int argc, char* argv[]) {
  int rank, msg;
  hipStream_t stream;

  // Initialize MPI
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  // Initialize NVSHMEM
  nvshmemx_init_attr_t attr;
  MPI_Comm comm = MPI_COMM_WORLD;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  // Initialize CUDA
  hipSetDevice(0);
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // Allocate message queue with NVSHMEM
  size_t rbuf_size = (1 << 29);
  ringbuf_t* rbuf = ringbuf_malloc(rbuf_size);
  size_t mbuf_size = (128);
  single_ringbuf_t* mbuf = single_ringbuf_malloc(mbuf_size);
  nvshmem_barrier_all();

  // Launch scheduler
  int grid_size = (argc > 1) ? atoi(argv[1]) : 1;
  int block_size = (argc > 2) ? atoi(argv[2]) : 1;
  if (!rank) {
    printf("NVCHARM\nGrid size: %d\nBlock size: %d\n", grid_size, block_size);
  }
  void* scheduler_args[4] = { &rbuf, &rbuf_size, &mbuf, &mbuf_size };
  nvshmemx_collective_launch((const void*)scheduler, grid_size, block_size,
      scheduler_args, 0, stream);
  cuda_check_error();
  hipStreamSynchronize(stream);
  //nvshmemx_barrier_all_on_stream(stream); // Hangs
  nvshmem_barrier_all();

  // Finalize NVSHMEM and MPI
  single_ringbuf_free(mbuf);
  ringbuf_free(rbuf);
  nvshmem_finalize();
  hipStreamDestroy(stream);
  MPI_Finalize();

  return 0;
}

template <typename T>
__device__ Chare<T>::Chare(T obj_, int n_chares_) : obj(obj_), n_chares(n_chares_) {
  /*
  // TODO: Create chare objects on all GPUs
  mapping = new Mapping[SM_CNT];
  int rem = n_chares % SM_CNT;
  int start_idx = 0;
  for (int i = 0; i < SM_CNT; i++) {
    int n_chares_sm = n_chares / SM_CNT;
    if (i < rem) n_chares_sm++;
    mapping[i].sm_id = i;
    mapping[i].start_idx = start_idx;
    mapping[i].end_idx = start_idx + n_chares_sm - 1;
    start_idx += n_chares_sm;

    //CreationMessage<T>* create_msg = new CreationMessage<T>(obj);
  }
  */
}

template <typename T>
__device__ void Chare<T>::invoke(int ep, int idx) {
  /*
  if (idx == -1) {
    // Broadcast to all chares
    for (int i = 0; i < n_chares; i++) {
      Message* msg = (Message*)malloc(sizeof(Message));
      msg->ep = ep;
      int target_sm = mapping[i];
      send(target_sm, msg);
    }
  } else {
    // P2P
    Message* msg = (Message*)malloc(sizeof(Message));
    msg->ep = ep;
    int target_sm = mapping[idx];
    send(target_sm, msg);
  }
  */
}
