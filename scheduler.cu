#include "hip/hip_runtime.h"
#include "scheduler.h"
#include "nvcharm.h"
#include "chare.h"
#include "ringbuf.h"

using namespace charm;

extern __device__ mpsc_ringbuf_t* rbuf;
extern __device__ size_t rbuf_size;
extern __device__ spsc_ringbuf_t* mbuf;
extern __device__ size_t mbuf_size;

extern __device__ chare_type* chare_types[];

__device__ envelope* charm::create_envelope(msgtype type, size_t msg_size) {
  // Secure region in my message pool
  ringbuf_off_t mret = spsc_ringbuf_acquire(mbuf, msg_size);
  assert(mret != -1 && mret < mbuf_size);

  // Create envelope
  envelope* env = new (mbuf->addr(mret)) envelope(type, msg_size, nvshmem_my_pe());

  return env;
}

__device__ void charm::send_msg(envelope* env, size_t msg_size, int dst_pe) {
  spsc_ringbuf_produce(mbuf);

  // Secure region in destination PE's message queue
  ringbuf_off_t rret;
  while ((rret = mpsc_ringbuf_acquire(rbuf, msg_size, dst_pe)) == -1) {}
  assert(rret < rbuf_size);

  // Send message
  printf("acquired %lld, sending to PE %d rbuf->addr(rret): %p, msg size %lld\n", rret, dst_pe, rbuf->addr(rret), env->size);
  nvshmem_char_put((char*)rbuf->addr(rret), (char*)env, env->size, dst_pe);
  nvshmem_quiet();
  mpsc_ringbuf_produce(rbuf, dst_pe);

  // Free region in my message pool
  size_t len, off;
  len = spsc_ringbuf_consume(mbuf, &off);
  spsc_ringbuf_release(mbuf, len);
}

__device__ void charm::send_reg_msg(int chare_id, int chare_idx, int ep_id,
                                    size_t payload_size, int dst_pe) {
  size_t msg_size = envelope::alloc_size(sizeof(charm::regular_msg) + payload_size);
  envelope* env = create_envelope(msgtype::regular, msg_size);

  charm::regular_msg* msg = new ((char*)env + sizeof(envelope)) charm::regular_msg(chare_id, chare_idx, ep_id);

  // TODO: Fill in payload

  send_msg(env, msg_size, dst_pe);
}

__device__ void charm::send_term_msg(int dst_pe) {
  size_t msg_size = envelope::alloc_size(0);
  envelope* env = create_envelope(msgtype::terminate, msg_size);

  send_msg(env, msg_size, dst_pe);
}

__device__ __forceinline__ ssize_t next_msg(void* addr, bool& term_flag) {
  envelope* env = (envelope*)addr;
#ifdef DEBUG
  printf("PE %d received msg type %d size %llu from PE %d\n",
         nvshmem_my_pe(), env->type, env->size, env->src_pe);
#endif

  if (env->type == msgtype::create) {
    // Creation message
    charm::create_msg* msg = (charm::create_msg*)((char*)env + sizeof(envelope));
#ifdef DEBUG
    printf("PE %d creation msg chare ID %d, start idx %d, end idx %d\n",
           nvshmem_my_pe(), msg->chare_id, msg->start_idx, msg->end_idx);
#endif
    charm::chare_type*& chare_type = chare_types[msg->chare_id];
    chare_type->alloc(msg->n_chares);
    void* packed_data = (char*)msg + sizeof(charm::create_msg);
    for (int i = 0; i < msg->n_chares; i++) {
      chare_type->unpack(packed_data, i);
    }
  } else if (env->type == msgtype::regular) {
    // Regular message
    charm::regular_msg* msg = (charm::regular_msg*)((char*)env + sizeof(envelope));
#ifdef DEBUG
    printf("PE %d regular msg chare ID %d chare idx %d EP ID %d\n", nvshmem_my_pe(), msg->chare_id, msg->chare_idx, msg->ep_id);
#endif
    charm::chare_type*& chare_type = chare_types[msg->chare_id];
    chare_type->call(msg->chare_idx, msg->ep_id);
  } else if (env->type == msgtype::terminate) {
    // Termination message
#ifdef DEBUG
    printf("PE %d terminate msg\n", nvshmem_my_pe());
#endif
    term_flag = true;
  }

  return env->size;
}

__device__ __forceinline__ void recv_msg(bool &term_flag) {
  size_t len, off;
  if ((len = mpsc_ringbuf_consume(rbuf, &off)) != 0) {
    // Retrieved a contiguous range, there could be multiple messages
    size_t rem = len;
    ssize_t ret;
    while (rem) {
      ret = next_msg(rbuf->addr(off), term_flag);
      off += ret;
      rem -= ret;
    }
    mpsc_ringbuf_release(rbuf, len);
  }
}

__global__ void charm::scheduler() {
  if (!blockIdx.x && !threadIdx.x) {
    bool term_flag = false;
    int my_pe = nvshmem_my_pe();
    int n_pes = nvshmem_n_pes();

    // Register all chares and entry methods
    register_chare_types(chare_types);

    // Initialize message queue
    mpsc_ringbuf_init(rbuf, rbuf_size);
    spsc_ringbuf_init(mbuf, mbuf_size);

    nvshmem_barrier_all();

    if (my_pe == 0) {
      // Execute user's main function
      main(chare_types);
    }

    nvshmem_barrier_all();

    // Receive messages and terminate
    do {
      recv_msg(term_flag);
    } while(!term_flag);
  }
}
